#include "hip/hip_runtime.h"
//
// Experimental Kernel for Kepler (Compute 3.5) devices
// code submitted by nVidia performance engineer Alexey Panteleev
// with modifications by Christian Buchner
//
// for Compute 3.5
// NOTE: compile this .cu module for compute_35,sm_35 with --maxrregcount=80
// for Compute 3.0
// NOTE: compile this .cu module for compute_30,sm_30 with --maxrregcount=63
//

#include <map>

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_runtime.h>

#include "miner.h"
#include "nv_kernel2.h"

#define THREADS_PER_WU 1  // single thread per hash 

#if __CUDA_ARCH__ < 350 
    // Kepler (Compute 3.0)
    #define __ldg(x) (*(x))
#endif

// grab lane ID
static __device__ __inline__ unsigned int __laneId() { unsigned int laneId; asm( "mov.u32 %0, %%laneid;" : "=r"( laneId ) ); return laneId; }

// forward references
template <int ALGO> __global__ void nv2_scrypt_core_kernelA(uint32_t *g_idata, int begin, int end);
template <int ALGO> __global__ void nv2_scrypt_core_kernelB(uint32_t *g_odata, int begin, int end);
template <int ALGO> __global__ void nv2_scrypt_core_kernelA_LG(uint32_t *g_idata, int begin, int end, unsigned int LOOKUP_GAP);
template <int ALGO> __global__ void nv2_scrypt_core_kernelB_LG(uint32_t *g_odata, int begin, int end, unsigned int LOOKUP_GAP);

// scratchbuf constants (pointers to scratch buffer for each work unit)
__constant__ uint32_t* c_V[TOTAL_WARP_LIMIT];

// iteration count N
__constant__ uint32_t c_N;
__constant__ uint32_t c_N_1; // N - 1
__constant__ uint32_t c_spacing; // (N+LOOKUP_GAP-1)/LOOKUP_GAP


NV2Kernel::NV2Kernel() : KernelInterface()
{
}

void NV2Kernel::set_scratchbuf_constants(int MAXWARPS, uint32_t** h_V)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_V), h_V, MAXWARPS*sizeof(uint32_t*), 0, hipMemcpyHostToDevice));
}

bool NV2Kernel::run_kernel(dim3 grid, dim3 threads, int WARPS_PER_BLOCK, int thr_id, hipStream_t stream, uint32_t* d_idata, uint32_t* d_odata, unsigned int N, unsigned int LOOKUP_GAP, bool interactive, bool benchmark, int texture_cache)
{
    bool success = true;

    // clear CUDA's error variable
    hipGetLastError();

    // make some constants available to kernel, update only initially and when changing
    static int prev_N[8] = {0,0,0,0,0,0,0,0};
    if (N != prev_N[thr_id]) {
        uint32_t h_N = N;
        checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N), &h_N, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream));
        uint32_t h_N_1 = N-1;
        checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_N_1), &h_N_1, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream));
        uint32_t h_spacing = (N+LOOKUP_GAP-1)/LOOKUP_GAP;
        checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(c_spacing), &h_spacing, sizeof(uint32_t), 0, hipMemcpyHostToDevice, stream));
        prev_N[thr_id] = N;
    }

    // First phase: Sequential writes to scratchpad.
    const int batch = device_batchsize[thr_id];
    const int sleeptime = 100;
    unsigned int pos = 0;
    int situation = 0;

    do 
    {
        if (LOOKUP_GAP == 1)
            switch(opt_algo) {
                case ALGO_SCRYPT:      nv2_scrypt_core_kernelA<ALGO_SCRYPT>     <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N)); break;
                case ALGO_SCRYPT_JANE: nv2_scrypt_core_kernelA<ALGO_SCRYPT_JANE><<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N)); break;
            }
        else
            switch(opt_algo) {
                case ALGO_SCRYPT:      nv2_scrypt_core_kernelA_LG<ALGO_SCRYPT>     <<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP); break;
                case ALGO_SCRYPT_JANE: nv2_scrypt_core_kernelA_LG<ALGO_SCRYPT_JANE><<< grid, threads, 0, stream >>>(d_idata, pos, min(pos+batch, N), LOOKUP_GAP); break;
            }
        
        if (!benchmark && interactive) {
            checkCudaErrors(MyStreamSynchronize(stream, ++situation, thr_id));
            usleep(sleeptime);
        }

        pos += batch;
    } while (pos < N);

    // Second phase: Random read access from scratchpad.
    pos = 0;
    do
    {
        if (pos > 0 && !benchmark && interactive) {
            checkCudaErrors(MyStreamSynchronize(stream, ++situation, thr_id));
            usleep(sleeptime);
        }

        if (LOOKUP_GAP == 1)
            switch(opt_algo) {
                case ALGO_SCRYPT:      nv2_scrypt_core_kernelB<ALGO_SCRYPT     ><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N)); break;
                case ALGO_SCRYPT_JANE: nv2_scrypt_core_kernelB<ALGO_SCRYPT_JANE><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N)); break;
            }
        else
            switch(opt_algo) {
                case ALGO_SCRYPT:      nv2_scrypt_core_kernelB_LG<ALGO_SCRYPT     ><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP); break;
                case ALGO_SCRYPT_JANE: nv2_scrypt_core_kernelB_LG<ALGO_SCRYPT_JANE><<< grid, threads, 0, stream >>>(d_odata, pos, min(pos+batch, N), LOOKUP_GAP); break;
            }

        pos += batch;
    } while (pos < N);

    // catch any kernel launch failures
    if (hipPeekAtLastError() != hipSuccess) success = false;

    return success;
}

static __device__ uint4& operator^=(uint4& left, const uint4& right)
{
    left.x ^= right.x;
    left.y ^= right.y;
    left.z ^= right.z;
    left.w ^= right.w;
    return left;
}

__device__ __forceinline__ uint4 __shfl(const uint4 val, unsigned int lane, unsigned int width)
{
    return make_uint4(
        (unsigned int)__shfl((int)val.x, lane, width),
        (unsigned int)__shfl((int)val.y, lane, width),
        (unsigned int)__shfl((int)val.z, lane, width),
        (unsigned int)__shfl((int)val.w, lane, width));
}

__device__ __forceinline__ void __transposed_write_BC(uint4 (&B)[4], uint4 (&C)[4], uint4 *D, int spacing)
{
    unsigned int laneId = __laneId();

    unsigned int lane8 = laneId%8;
    unsigned int tile  = laneId/8;
    
    uint4 T1[8], T2[8];

    /* Source matrix, A-H are threads, 0-7 are data items, thread A is marked with `*`:

       *A0  B0  C0  D0  E0  F0  G0  H0
       *A1  B1  C1  D1  E1  F1  G1  H1
       *A2  B2  C2  D2  E2  F2  G2  H2
       *A3  B3  C3  D3  E3  F3  G3  H3
       *A4  B4  C4  D4  E4  F4  G4  H4
       *A5  B5  C5  D5  E5  F5  G5  H5
       *A6  B6  C6  D6  E6  F6  G6  H6
       *A7  B7  C7  D7  E7  F7  G7  H7
    */

    // rotate rows
    T1[0] = B[0];
    T1[1] = __shfl(B[1], lane8 + 7, 8);
    T1[2] = __shfl(B[2], lane8 + 6, 8);
    T1[3] = __shfl(B[3], lane8 + 5, 8);
    T1[4] = __shfl(C[0], lane8 + 4, 8);
    T1[5] = __shfl(C[1], lane8 + 3, 8);
    T1[6] = __shfl(C[2], lane8 + 2, 8);
    T1[7] = __shfl(C[3], lane8 + 1, 8);

    /* Matrix after row rotates:

       *A0  B0  C0  D0  E0  F0  G0  H0
        H1 *A1  B1  C1  D1  E1  F1  G1
        G2  H2 *A2  B2  C2  D2  E2  F2
        F3  G3  H3 *A3  B3  C3  D3  E3
        E4  F4  G4  H4 *A4  B4  C4  D4
        D5  E5  F5  G5  H5 *A5  B5  C5
        C6  D6  E6  F6  G6  H6 *A6  B6
        B7  C7  D7  E7  F7  G7  H7 *A7
    */

    // rotate columns up using a barrel shifter simulation
    // column X is rotated up by (X+1) items
#pragma unroll 8
    for(int n = 0; n < 8; n++) T2[n] = ((lane8+1) & 1) ? T1[(n+1) % 8] : T1[n];
#pragma unroll 8
    for(int n = 0; n < 8; n++) T1[n] = ((lane8+1) & 2) ? T2[(n+2) % 8] : T2[n];
#pragma unroll 8
    for(int n = 0; n < 8; n++) T2[n] = ((lane8+1) & 4) ? T1[(n+4) % 8] : T1[n];

    /* Matrix after column rotates:

        H1  H2  H3  H4  H5  H6  H7  H0
        G2  G3  G4  G5  G6  G7  G0  G1   
        F3  F4  F5  F6  F7  F0  F1  F2       
        E4  E5  E6  E7  E0  E1  E2  E3           
        D5  D6  D7  D0  D1  D2  D3  D4               
        C6  C7  C0  C1  C2  C3  C4  C5                   
        B7  B0  B1  B2  B3  B4  B5  B6                       
       *A0 *A1 *A2 *A3 *A4 *A5 *A6 *A7
    */

    // rotate rows again using address math and write to D, in reverse row order
    D[spacing*2*(32*tile   )+ lane8     ] = T2[7];
    D[spacing*2*(32*tile+4 )+(lane8+7)%8] = T2[6];
    D[spacing*2*(32*tile+8 )+(lane8+6)%8] = T2[5];
    D[spacing*2*(32*tile+12)+(lane8+5)%8] = T2[4];
    D[spacing*2*(32*tile+16)+(lane8+4)%8] = T2[3];
    D[spacing*2*(32*tile+20)+(lane8+3)%8] = T2[2];
    D[spacing*2*(32*tile+24)+(lane8+2)%8] = T2[1];
    D[spacing*2*(32*tile+28)+(lane8+1)%8] = T2[0];
}

__device__ __forceinline__ void __transposed_read_BC(const uint4 *S, uint4 (&B)[4], uint4 (&C)[4], int spacing, int row)
{
    unsigned int laneId = __laneId();

    unsigned int lane8 = laneId%8;
    unsigned int tile  = laneId/8;

    // Perform the same transposition as in __transposed_write_BC, but in reverse order.
    // See the illustrations in comments for __transposed_write_BC.

    // read and rotate rows, in reverse row order
    uint4 T1[8], T2[8];
    T1[7] = __ldg(&S[(spacing*2*(32*tile   ) +  lane8      + 8*__shfl(row, 0, 8))]);
    T1[6] = __ldg(&S[(spacing*2*(32*tile+4 ) + (lane8+7)%8 + 8*__shfl(row, 1, 8))]);
    T1[5] = __ldg(&S[(spacing*2*(32*tile+8 ) + (lane8+6)%8 + 8*__shfl(row, 2, 8))]);
    T1[4] = __ldg(&S[(spacing*2*(32*tile+12) + (lane8+5)%8 + 8*__shfl(row, 3, 8))]);
    T1[3] = __ldg(&S[(spacing*2*(32*tile+16) + (lane8+4)%8 + 8*__shfl(row, 4, 8))]);
    T1[2] = __ldg(&S[(spacing*2*(32*tile+20) + (lane8+3)%8 + 8*__shfl(row, 5, 8))]);
    T1[1] = __ldg(&S[(spacing*2*(32*tile+24) + (lane8+2)%8 + 8*__shfl(row, 6, 8))]);
    T1[0] = __ldg(&S[(spacing*2*(32*tile+28) + (lane8+1)%8 + 8*__shfl(row, 7, 8))]);
    
    // rotate columns down using a barrel shifter simulation
    // column X is rotated down by (X+1) items, or up by (8-(X+1)) = (7-X) items
#pragma unroll 8
    for(int n = 0; n < 8; n++) T2[n] = ((7-lane8) & 1) ? T1[(n+1) % 8] : T1[n];
#pragma unroll 8
    for(int n = 0; n < 8; n++) T1[n] = ((7-lane8) & 2) ? T2[(n+2) % 8] : T2[n];
#pragma unroll 8
    for(int n = 0; n < 8; n++) T2[n] = ((7-lane8) & 4) ? T1[(n+4) % 8] : T1[n];
    
    // rotate rows
    B[0] = T2[0];
    B[1] = __shfl(T2[1], lane8 + 1, 8);
    B[2] = __shfl(T2[2], lane8 + 2, 8);
    B[3] = __shfl(T2[3], lane8 + 3, 8);
    C[0] = __shfl(T2[4], lane8 + 4, 8);
    C[1] = __shfl(T2[5], lane8 + 5, 8);
    C[2] = __shfl(T2[6], lane8 + 6, 8);
    C[3] = __shfl(T2[7], lane8 + 7, 8);

}

__device__ __forceinline__ void __transposed_xor_BC(const uint4 *S, uint4 (&B)[4], uint4 (&C)[4], int spacing, int row)
{
    uint4 BT[4], CT[4];
    __transposed_read_BC(S, BT, CT, spacing, row);

#pragma unroll 4
    for(int n = 0; n < 4; n++) 
    {
        B[n] ^= BT[n];
        C[n] ^= CT[n];
    }
}

#if __CUDA_ARCH__ < 350 
    // Kepler (Compute 3.0)
    #define ROTL(a, b) ((a)<<(b))|((a)>>(32-(b)))
#else
    // Kepler (Compute 3.5)
    #define ROTL(a, b) __funnelshift_l( a, a, b );
#endif



#if 0

#define QUARTER(a,b,c,d) \
    a += b; d ^= a; d = ROTL(d,16); \
    c += d; b ^= c; b = ROTL(b,12); \
    a += b; d ^= a; d = ROTL(d,8); \
    c += d; b ^= c; b = ROTL(b,7);

static __device__ void xor_chacha8(uint4 *B, uint4 *C)
{
    uint32_t x[16];
    x[0]=(B[0].x ^= C[0].x);
    x[1]=(B[0].y ^= C[0].y);
    x[2]=(B[0].z ^= C[0].z);
    x[3]=(B[0].w ^= C[0].w);
    x[4]=(B[1].x ^= C[1].x);
    x[5]=(B[1].y ^= C[1].y);
    x[6]=(B[1].z ^= C[1].z);
    x[7]=(B[1].w ^= C[1].w);
    x[8]=(B[2].x ^= C[2].x);
    x[9]=(B[2].y ^= C[2].y);
    x[10]=(B[2].z ^= C[2].z);
    x[11]=(B[2].w ^= C[2].w);
    x[12]=(B[3].x ^= C[3].x);
    x[13]=(B[3].y ^= C[3].y);
    x[14]=(B[3].z ^= C[3].z);
    x[15]=(B[3].w ^= C[3].w);

    /* Operate on columns. */
    QUARTER( x[0], x[4], x[ 8], x[12] )
    QUARTER( x[1], x[5], x[ 9], x[13] )
    QUARTER( x[2], x[6], x[10], x[14] )
    QUARTER( x[3], x[7], x[11], x[15] )

    /* Operate on diagonals */
    QUARTER( x[0], x[5], x[10], x[15] )
    QUARTER( x[1], x[6], x[11], x[12] )
    QUARTER( x[2], x[7], x[ 8], x[13] )
    QUARTER( x[3], x[4], x[ 9], x[14] )

    /* Operate on columns. */
    QUARTER( x[0], x[4], x[ 8], x[12] )
    QUARTER( x[1], x[5], x[ 9], x[13] )
    QUARTER( x[2], x[6], x[10], x[14] )
    QUARTER( x[3], x[7], x[11], x[15] )

    /* Operate on diagonals */
    QUARTER( x[0], x[5], x[10], x[15] )
    QUARTER( x[1], x[6], x[11], x[12] )
    QUARTER( x[2], x[7], x[ 8], x[13] )
    QUARTER( x[3], x[4], x[ 9], x[14] )

    /* Operate on columns. */
    QUARTER( x[0], x[4], x[ 8], x[12] )
    QUARTER( x[1], x[5], x[ 9], x[13] )
    QUARTER( x[2], x[6], x[10], x[14] )
    QUARTER( x[3], x[7], x[11], x[15] )

    /* Operate on diagonals */
    QUARTER( x[0], x[5], x[10], x[15] )
    QUARTER( x[1], x[6], x[11], x[12] )
    QUARTER( x[2], x[7], x[ 8], x[13] )
    QUARTER( x[3], x[4], x[ 9], x[14] )

    /* Operate on columns. */
    QUARTER( x[0], x[4], x[ 8], x[12] )
    QUARTER( x[1], x[5], x[ 9], x[13] )
    QUARTER( x[2], x[6], x[10], x[14] )
    QUARTER( x[3], x[7], x[11], x[15] )

    /* Operate on diagonals */
    QUARTER( x[0], x[5], x[10], x[15] )
    QUARTER( x[1], x[6], x[11], x[12] )
    QUARTER( x[2], x[7], x[ 8], x[13] )
    QUARTER( x[3], x[4], x[ 9], x[14] )

    B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
    B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

#else

#define ADD4(d1,d2,d3,d4,s1,s2,s3,s4) \
    d1 += s1; d2 += s2; d3 += s3; d4 += s4;

#define XOR4(d1,d2,d3,d4,s1,s2,s3,s4) \
    d1 ^= s1; d2 ^= s2; d3 ^= s3; d4 ^= s4;

#define ROTL4(d1,d2,d3,d4,amt) \
    d1 = ROTL(d1, amt); d2 = ROTL(d2, amt); d3 = ROTL(d3, amt); d4 = ROTL(d4, amt);

#define QROUND(a1,a2,a3,a4, b1,b2,b3,b4, c1,c2,c3,c4, amt) \
    ADD4 (a1,a2,a3,a4, c1,c2,c3,c4) \
    XOR4 (b1,b2,b3,b4, a1,a2,a3,a4) \
    ROTL4(b1,b2,b3,b4, amt)

static __device__ void xor_chacha8(uint4 *B, uint4 *C)
{
    uint32_t x[16];
    x[0]=(B[0].x ^= C[0].x);
    x[1]=(B[0].y ^= C[0].y);
    x[2]=(B[0].z ^= C[0].z);
    x[3]=(B[0].w ^= C[0].w);
    x[4]=(B[1].x ^= C[1].x);
    x[5]=(B[1].y ^= C[1].y);
    x[6]=(B[1].z ^= C[1].z);
    x[7]=(B[1].w ^= C[1].w);
    x[8]=(B[2].x ^= C[2].x);
    x[9]=(B[2].y ^= C[2].y);
    x[10]=(B[2].z ^= C[2].z);
    x[11]=(B[2].w ^= C[2].w);
    x[12]=(B[3].x ^= C[3].x);
    x[13]=(B[3].y ^= C[3].y);
    x[14]=(B[3].z ^= C[3].z);
    x[15]=(B[3].w ^= C[3].w);

    /* Operate on columns. */
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
    QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
    QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

    /* Operate on diagonals */
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
    QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
    QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

    /* Operate on columns. */
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
    QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
    QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

    /* Operate on diagonals */
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
    QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
    QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

    /* Operate on columns. */
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
    QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
    QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

    /* Operate on diagonals */
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
    QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
    QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

    /* Operate on columns. */
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7], 16);
    QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15], 12);
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[12],x[13],x[14],x[15], x[ 4],x[ 5],x[ 6],x[ 7],  8);
    QROUND(x[ 8],x[ 9],x[10],x[11], x[ 4],x[ 5],x[ 6],x[ 7], x[12],x[13],x[14],x[15],  7);

    /* Operate on diagonals */
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4], 16);
    QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14], 12);
    QROUND(x[ 0],x[ 1],x[ 2],x[ 3], x[15],x[12],x[13],x[14], x[ 5],x[ 6],x[ 7],x[ 4],  8);
    QROUND(x[10],x[11],x[ 8],x[ 9], x[ 5],x[ 6],x[ 7],x[ 4], x[15],x[12],x[13],x[14],  7);

    B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
    B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}

#endif


#define ROTL7(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 7); a1^=ROTL(a10, 7); a2^=ROTL(a20, 7); a3^=ROTL(a30, 7);\
};\

#define ROTL9(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 9); a1^=ROTL(a10, 9); a2^=ROTL(a20, 9); a3^=ROTL(a30, 9);\
};\

#define ROTL13(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 13); a1^=ROTL(a10, 13); a2^=ROTL(a20, 13); a3^=ROTL(a30, 13);\
};\

#define ROTL18(a0,a1,a2,a3,a00,a10,a20,a30){\
a0^=ROTL(a00, 18); a1^=ROTL(a10, 18); a2^=ROTL(a20, 18); a3^=ROTL(a30, 18);\
};\

static __device__ void xor_salsa8(uint4 *B, uint4 *C)
{
    uint32_t x[16];
    x[0]=(B[0].x ^= C[0].x);
    x[1]=(B[0].y ^= C[0].y);
    x[2]=(B[0].z ^= C[0].z);
    x[3]=(B[0].w ^= C[0].w);
    x[4]=(B[1].x ^= C[1].x);
    x[5]=(B[1].y ^= C[1].y);
    x[6]=(B[1].z ^= C[1].z);
    x[7]=(B[1].w ^= C[1].w);
    x[8]=(B[2].x ^= C[2].x);
    x[9]=(B[2].y ^= C[2].y);
    x[10]=(B[2].z ^= C[2].z);
    x[11]=(B[2].w ^= C[2].w);
    x[12]=(B[3].x ^= C[3].x);
    x[13]=(B[3].y ^= C[3].y);
    x[14]=(B[3].z ^= C[3].z);
    x[15]=(B[3].w ^= C[3].w);

    /* Operate on columns. */
    ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
    ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
    ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
    ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
    ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
    ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
    ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
    ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
    ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
    ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
    ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
    ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
    ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
    ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
    ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
    ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
    ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
    ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
    ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
    ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
    ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
    ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
    ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
    ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    /* Operate on columns. */
    ROTL7(x[4],x[9],x[14],x[3],x[0]+x[12],x[1]+x[5],x[6]+x[10],x[11]+x[15]);
    ROTL9(x[8],x[13],x[2],x[7],x[0]+x[4],x[5]+x[9],x[10]+x[14],x[3]+x[15]);
    ROTL13(x[12],x[1],x[6],x[11],x[4]+x[8],x[9]+x[13],x[2]+x[14],x[3]+x[7]);
    ROTL18(x[0],x[5],x[10],x[15],x[8]+x[12],x[1]+x[13],x[2]+x[6],x[7]+x[11]);

    /* Operate on rows. */
    ROTL7(x[1],x[6],x[11],x[12],x[0]+x[3],x[4]+x[5],x[9]+x[10],x[14]+x[15]);
    ROTL9(x[2],x[7],x[8],x[13],x[0]+x[1],x[5]+x[6],x[10]+x[11],x[12]+x[15]);
    ROTL13(x[3],x[4],x[9],x[14],x[1]+x[2],x[6]+x[7],x[8]+x[11],x[12]+x[13]);
    ROTL18(x[0],x[5],x[10],x[15],x[2]+x[3],x[4]+x[7],x[8]+x[9],x[13]+x[14]);

    B[0].x += x[0]; B[0].y += x[1]; B[0].z += x[2];  B[0].w += x[3];  B[1].x += x[4];  B[1].y += x[5];  B[1].z += x[6];  B[1].w += x[7];
    B[2].x += x[8]; B[2].y += x[9]; B[2].z += x[10]; B[2].w += x[11]; B[3].x += x[12]; B[3].y += x[13]; B[3].z += x[14]; B[3].w += x[15];
}


template <int ALGO> static __device__ void block_mixer(uint4 *B, uint4 *C)
{
  switch (ALGO)
  {
    case ALGO_SCRYPT:      xor_salsa8(B, C); break;
    case ALGO_SCRYPT_JANE: xor_chacha8(B, C); break;
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Experimental Scrypt core kernel for Titan devices.
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <int ALGO> __global__ void nv2_scrypt_core_kernelA(uint32_t *g_idata, int begin, int end)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
    g_idata += 32 * offset;
    uint32_t * V = c_V[offset / warpSize];
    uint4 B[4], C[4];
    int i = begin;

    if(i == 0) {
        __transposed_read_BC((uint4*)g_idata, B, C, 1, 0);
        __transposed_write_BC(B, C, (uint4*)V, c_N); 
        ++i;
    } else
        __transposed_read_BC((uint4*)(V + (i-1)*32), B, C, c_N, 0);

    while(i < end) {
        block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
        __transposed_write_BC(B, C, (uint4*)(V + i*32), c_N); 
        ++i;
    }
}

template <int ALGO> __global__ void nv2_scrypt_core_kernelA_LG(uint32_t *g_idata, int begin, int end, unsigned int LOOKUP_GAP)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
    g_idata += 32 * offset;
    uint32_t * V = c_V[offset / warpSize];
    uint4 B[4], C[4];
    int i = begin;

    if(i == 0) {
        __transposed_read_BC((uint4*)g_idata, B, C, 1, 0);
        __transposed_write_BC(B, C, (uint4*)V, c_spacing); 
        ++i;
    } else {
        int pos = (i-1)/LOOKUP_GAP, loop = (i-1)-pos*LOOKUP_GAP;
        __transposed_read_BC((uint4*)(V + pos*32), B, C, c_spacing, 0);
        while(loop--) { block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B); }
    }

    while(i < end) {
        block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
        if (i % LOOKUP_GAP == 0)
          __transposed_write_BC(B, C, (uint4*)(V + (i/LOOKUP_GAP)*32), c_spacing); 
        ++i;
    }
}

template <int ALGO> __global__ void nv2_scrypt_core_kernelB(uint32_t *g_odata, int begin, int end)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
    g_odata += 32 * offset;
    uint32_t * V = c_V[offset / warpSize];
    uint4 B[4], C[4];

    if(begin == 0) {
        __transposed_read_BC((uint4*)V, B, C, c_N, c_N_1);
        block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
    } else
        __transposed_read_BC((uint4*)g_odata, B, C, 1, 0);

    for (int i = begin; i < end; i++)  {
        int slot = C[0].x & c_N_1;
        __transposed_xor_BC((uint4*)(V), B, C, c_N, slot);
        block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
    }

    __transposed_write_BC(B, C, (uint4*)(g_odata), 1);
}

template <int ALGO> __global__ void nv2_scrypt_core_kernelB_LG(uint32_t *g_odata, int begin, int end, unsigned int LOOKUP_GAP)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x / warpSize * warpSize;
    g_odata += 32 * offset;
    uint32_t * V = c_V[offset / warpSize];
    uint4 B[4], C[4];

    if(begin == 0) {
      int pos = c_N_1/LOOKUP_GAP, loop = 1 + (c_N_1-pos*LOOKUP_GAP);
      __transposed_read_BC((uint4*)V, B, C, c_spacing, pos);
      while(loop--) { block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B); }
    } else {
        __transposed_read_BC((uint4*)g_odata, B, C, 1, 0);
    }

    for (int i = begin; i < end; i++)  {
        int slot = C[0].x & c_N_1;
        int pos = slot/LOOKUP_GAP, loop = slot-pos*LOOKUP_GAP;
        uint4 b[4], c[4]; __transposed_read_BC((uint4*)(V), b, c, c_spacing, pos);
        while(loop--) { block_mixer<ALGO>(b, c); block_mixer<ALGO>(c, b); }
#pragma unroll 4
        for(int n = 0; n < 4; n++) { B[n] ^= b[n]; C[n] ^= c[n]; }
        block_mixer<ALGO>(B, C); block_mixer<ALGO>(C, B);
    }

    __transposed_write_BC(B, C, (uint4*)(g_odata), 1);
}


//
// Maxcoin related Keccak implementation (Keccak256)
//

// from salsa_kernel.cu
extern std::map<int, int> context_blocks;
extern std::map<int, int> context_wpb;
extern std::map<int, KernelInterface *> context_kernel;
extern std::map<int, hipStream_t> context_streams[2];
extern std::map<int, uint32_t *> context_hash[2];

__constant__ uint64_t ptarget64[4];

#define ROL(a, offset) ((((uint64_t)a) << ((offset) % 64)) ^ (((uint64_t)a) >> (64-((offset) % 64))))
#define ROL_mult8(a, offset) ROL(a, offset)

__constant__ uint64_t KeccakF_RoundConstants[24];

static uint64_t host_KeccakF_RoundConstants[24] = 
{
    (uint64_t)0x0000000000000001ULL,
    (uint64_t)0x0000000000008082ULL,
    (uint64_t)0x800000000000808aULL,
    (uint64_t)0x8000000080008000ULL,
    (uint64_t)0x000000000000808bULL,
    (uint64_t)0x0000000080000001ULL,
    (uint64_t)0x8000000080008081ULL,
    (uint64_t)0x8000000000008009ULL,
    (uint64_t)0x000000000000008aULL,
    (uint64_t)0x0000000000000088ULL,
    (uint64_t)0x0000000080008009ULL,
    (uint64_t)0x000000008000000aULL,
    (uint64_t)0x000000008000808bULL,
    (uint64_t)0x800000000000008bULL,
    (uint64_t)0x8000000000008089ULL,
    (uint64_t)0x8000000000008003ULL,
    (uint64_t)0x8000000000008002ULL,
    (uint64_t)0x8000000000000080ULL,
    (uint64_t)0x000000000000800aULL,
    (uint64_t)0x800000008000000aULL,
    (uint64_t)0x8000000080008081ULL,
    (uint64_t)0x8000000000008080ULL,
    (uint64_t)0x0000000080000001ULL,
    (uint64_t)0x8000000080008008ULL
};

__constant__ uint64_t pdata64[10];

static __device__ uint32_t cuda_swab32(uint32_t x)
{
    return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
          | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
}

__global__ void titan_crypto_hash( uint64_t *g_out, uint32_t nonce, uint32_t *g_good, bool validate )
{
    uint64_t Aba, Abe, Abi, Abo, Abu;
    uint64_t Aga, Age, Agi, Ago, Agu;
    uint64_t Aka, Ake, Aki, Ako, Aku;
    uint64_t Ama, Ame, Ami, Amo, Amu;
    uint64_t Asa, Ase, Asi, Aso, Asu;
    uint64_t BCa, BCe, BCi, BCo, BCu;
    uint64_t Da, De, Di, Do, Du;
    uint64_t Eba, Ebe, Ebi, Ebo, Ebu;
    uint64_t Ega, Ege, Egi, Ego, Egu;
    uint64_t Eka, Eke, Eki, Eko, Eku;
    uint64_t Ema, Eme, Emi, Emo, Emu;
    uint64_t Esa, Ese, Esi, Eso, Esu;

    //copyFromState(A, state)
    Aba = pdata64[0];
    Abe = pdata64[1];
    Abi = pdata64[2];
    Abo = pdata64[3];
    Abu = pdata64[4];
    Aga = pdata64[5];
    Age = pdata64[6];
    Agi = pdata64[7];
    Ago = pdata64[8];
    Agu = (pdata64[9] & 0x00000000FFFFFFFFULL) | (((uint64_t)cuda_swab32(nonce + ((blockIdx.x * blockDim.x) + threadIdx.x))) << 32);
    Aka = 0x0000000000000001ULL;
    Ake = 0;
    Aki = 0;
    Ako = 0;
    Aku = 0;
    Ama = 0;
    Ame = 0x8000000000000000ULL;
    Ami = 0;
    Amo = 0;
    Amu = 0;
    Asa = 0;
    Ase = 0;
    Asi = 0;
    Aso = 0;
    Asu = 0;

#pragma unroll 12
    for( int laneCount = 0; laneCount < 24; laneCount += 2 )
    {
        //    prepareTheta
        BCa = Aba^Aga^Aka^Ama^Asa;
        BCe = Abe^Age^Ake^Ame^Ase;
        BCi = Abi^Agi^Aki^Ami^Asi;
        BCo = Abo^Ago^Ako^Amo^Aso;
        BCu = Abu^Agu^Aku^Amu^Asu;

        //thetaRhoPiChiIotaPrepareTheta(round  , A, E)
        Da = BCu^ROL(BCe, 1);
        De = BCa^ROL(BCi, 1);
        Di = BCe^ROL(BCo, 1);
        Do = BCi^ROL(BCu, 1);
        Du = BCo^ROL(BCa, 1);

        Aba ^= Da;
        BCa = Aba;
        Age ^= De;
        BCe = ROL(Age, 44);
        Aki ^= Di;
        BCi = ROL(Aki, 43);
        Amo ^= Do;
        BCo = ROL(Amo, 21);
        Asu ^= Du;
        BCu = ROL(Asu, 14);
        Eba =   BCa ^((~BCe)&  BCi );
        Eba ^= (uint64_t)KeccakF_RoundConstants[laneCount];
        Ebe =   BCe ^((~BCi)&  BCo );
        Ebi =   BCi ^((~BCo)&  BCu );
        Ebo =   BCo ^((~BCu)&  BCa );
        Ebu =   BCu ^((~BCa)&  BCe );

        Abo ^= Do;
        BCa = ROL(Abo, 28);
        Agu ^= Du;
        BCe = ROL(Agu, 20);
        Aka ^= Da;
        BCi = ROL(Aka,  3);
        Ame ^= De;
        BCo = ROL(Ame, 45);
        Asi ^= Di;
        BCu = ROL(Asi, 61);
        Ega =   BCa ^((~BCe)&  BCi );
        Ege =   BCe ^((~BCi)&  BCo );
        Egi =   BCi ^((~BCo)&  BCu );
        Ego =   BCo ^((~BCu)&  BCa );
        Egu =   BCu ^((~BCa)&  BCe );

        Abe ^= De;
        BCa = ROL(Abe,  1);
        Agi ^= Di;
        BCe = ROL(Agi,  6);
        Ako ^= Do;
        BCi = ROL(Ako, 25);
        Amu ^= Du;
        BCo = ROL_mult8(Amu,  8);
        Asa ^= Da;
        BCu = ROL(Asa, 18);
        Eka =   BCa ^((~BCe)&  BCi );
        Eke =   BCe ^((~BCi)&  BCo );
        Eki =   BCi ^((~BCo)&  BCu );
        Eko =   BCo ^((~BCu)&  BCa );
        Eku =   BCu ^((~BCa)&  BCe );

        Abu ^= Du;
        BCa = ROL(Abu, 27);
        Aga ^= Da;
        BCe = ROL(Aga, 36);
        Ake ^= De;
        BCi = ROL(Ake, 10);
        Ami ^= Di;
        BCo = ROL(Ami, 15);
        Aso ^= Do;
        BCu = ROL_mult8(Aso, 56);
        Ema =   BCa ^((~BCe)&  BCi );
        Eme =   BCe ^((~BCi)&  BCo );
        Emi =   BCi ^((~BCo)&  BCu );
        Emo =   BCo ^((~BCu)&  BCa );
        Emu =   BCu ^((~BCa)&  BCe );

        Abi ^= Di;
        BCa = ROL(Abi, 62);
        Ago ^= Do;
        BCe = ROL(Ago, 55);
        Aku ^= Du;
        BCi = ROL(Aku, 39);
        Ama ^= Da;
        BCo = ROL(Ama, 41);
        Ase ^= De;
        BCu = ROL(Ase,  2);
        Esa =   BCa ^((~BCe)&  BCi );
        Ese =   BCe ^((~BCi)&  BCo );
        Esi =   BCi ^((~BCo)&  BCu );
        Eso =   BCo ^((~BCu)&  BCa );
        Esu =   BCu ^((~BCa)&  BCe );

        //    prepareTheta
        BCa = Eba^Ega^Eka^Ema^Esa;
        BCe = Ebe^Ege^Eke^Eme^Ese;
        BCi = Ebi^Egi^Eki^Emi^Esi;
        BCo = Ebo^Ego^Eko^Emo^Eso;
        BCu = Ebu^Egu^Eku^Emu^Esu;

        //thetaRhoPiChiIotaPrepareTheta(round+1, E, A)
        Da = BCu^ROL(BCe, 1);
        De = BCa^ROL(BCi, 1);
        Di = BCe^ROL(BCo, 1);
        Do = BCi^ROL(BCu, 1);
        Du = BCo^ROL(BCa, 1);

        Eba ^= Da;
        BCa = Eba;
        Ege ^= De;
        BCe = ROL(Ege, 44);
        Eki ^= Di;
        BCi = ROL(Eki, 43);
        Emo ^= Do;
        BCo = ROL(Emo, 21);
        Esu ^= Du;
        BCu = ROL(Esu, 14);
        Aba =   BCa ^((~BCe)&  BCi );
        Aba ^= (uint64_t)KeccakF_RoundConstants[laneCount+1];
        Abe =   BCe ^((~BCi)&  BCo );
        Abi =   BCi ^((~BCo)&  BCu );
        Abo =   BCo ^((~BCu)&  BCa );
        Abu =   BCu ^((~BCa)&  BCe );

        Ebo ^= Do;
        BCa = ROL(Ebo, 28);
        Egu ^= Du;
        BCe = ROL(Egu, 20);
        Eka ^= Da;
        BCi = ROL(Eka, 3);
        Eme ^= De;
        BCo = ROL(Eme, 45);
        Esi ^= Di;
        BCu = ROL(Esi, 61);
        Aga =   BCa ^((~BCe)&  BCi );
        Age =   BCe ^((~BCi)&  BCo );
        Agi =   BCi ^((~BCo)&  BCu );
        Ago =   BCo ^((~BCu)&  BCa );
        Agu =   BCu ^((~BCa)&  BCe );

        Ebe ^= De;
        BCa = ROL(Ebe, 1);
        Egi ^= Di;
        BCe = ROL(Egi, 6);
        Eko ^= Do;
        BCi = ROL(Eko, 25);
        Emu ^= Du;
        BCo = ROL_mult8(Emu, 8);
        Esa ^= Da;
        BCu = ROL(Esa, 18);
        Aka =   BCa ^((~BCe)&  BCi );
        Ake =   BCe ^((~BCi)&  BCo );
        Aki =   BCi ^((~BCo)&  BCu );
        Ako =   BCo ^((~BCu)&  BCa );
        Aku =   BCu ^((~BCa)&  BCe );

        Ebu ^= Du;
        BCa = ROL(Ebu, 27);
        Ega ^= Da;
        BCe = ROL(Ega, 36);
        Eke ^= De;
        BCi = ROL(Eke, 10);
        Emi ^= Di;
        BCo = ROL(Emi, 15);
        Eso ^= Do;
        BCu = ROL_mult8(Eso, 56);
        Ama =   BCa ^((~BCe)&  BCi );
        Ame =   BCe ^((~BCi)&  BCo );
        Ami =   BCi ^((~BCo)&  BCu );
        Amo =   BCo ^((~BCu)&  BCa );
        Amu =   BCu ^((~BCa)&  BCe );

        Ebi ^= Di;
        BCa = ROL(Ebi, 62);
        Ego ^= Do;
        BCe = ROL(Ego, 55);
        Eku ^= Du;
        BCi = ROL(Eku, 39);
        Ema ^= Da;
        BCo = ROL(Ema, 41);
        Ese ^= De;
        BCu = ROL(Ese, 2);
        Asa =   BCa ^((~BCe)&  BCi );
        Ase =   BCe ^((~BCi)&  BCo );
        Asi =   BCi ^((~BCo)&  BCu );
        Aso =   BCo ^((~BCu)&  BCa );
        Asu =   BCu ^((~BCa)&  BCe );
    }

    if (validate) {
        g_out += 4 * ((blockIdx.x * blockDim.x) + threadIdx.x);
        g_out[3] = Abo;
        g_out[2] = Abi;
        g_out[1] = Abe;
        g_out[0] = Aba;
    }
    
    // the likelyhood of meeting the hashing target is so low, that we're not guarding this
    // with atomic writes, locks or similar...
    uint64_t *g_good64 = (uint64_t*)g_good;
    if (Abo <=  ptarget64[3]) {
        if (Abo < g_good64[3]) {
            g_good64[3] = Abo;
            g_good64[2] = Abi;
            g_good64[1] = Abe;
            g_good64[0] = Aba;
            g_good[8] = nonce + ((blockIdx.x * blockDim.x) + threadIdx.x);
        }
    }
}

static std::map<int, uint32_t *> context_good[2];

void NV2Kernel::prepare_keccak256(int thr_id, const uint32_t host_pdata[20], const uint32_t host_ptarget[8])
{
    static bool init[8] = {false, false, false, false, false, false, false, false};
    if (!init[thr_id])
    {
        hipMemcpyToSymbol(HIP_SYMBOL(KeccakF_RoundConstants), host_KeccakF_RoundConstants, sizeof(host_KeccakF_RoundConstants), 0, hipMemcpyHostToDevice);

        // allocate pinned host memory for good hashes
        uint32_t *tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[0][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[1][thr_id] = tmp;

        init[thr_id] = true;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(pdata64), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(ptarget64), host_ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}

uint32_t NV2Kernel::do_keccak256(int thr_id, int stream, uint32_t *hash, uint32_t nonce, int throughput, bool do_d2h)
{
    uint32_t result = 0xffffffff;
  
    unsigned int GRID_BLOCKS = context_blocks[thr_id];
    unsigned int WARPS_PER_BLOCK = context_wpb[thr_id];

    // setup execution parameters
    dim3  grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
    dim3  threads(THREADS_PER_WU*WU_PER_BLOCK, 1, 1);

    checkCudaErrors(hipMemsetAsync(context_good[stream][thr_id], 0xff, 9 * sizeof(uint32_t), context_streams[stream][thr_id]));

    titan_crypto_hash<<<grid, threads, 0, context_streams[stream][thr_id]>>>((uint64_t*)context_hash[stream][thr_id], nonce, context_good[stream][thr_id], do_d2h);

    // copy hashes from device memory to host (ALL hashes, lots of data...)
    if (do_d2h) {
        size_t mem_size = throughput * sizeof(uint32_t) * 8;
        checkCudaErrors(hipMemcpyAsync(hash, context_hash[stream][thr_id], mem_size,
                        hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
    }
    else {
        // asynchronous copy of winning nonce (just 4 bytes...)
        checkCudaErrors(hipMemcpyAsync(hash, context_good[stream][thr_id]+8, sizeof(uint32_t),
                        hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
    }

    return result;
}
