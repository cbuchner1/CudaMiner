#include "hip/hip_runtime.h"
//
//  =============== KECCAK part on nVidia GPU ======================
//
// NOTE: compile this .cu module for compute_10,sm_10 with --maxrregcount=64
//
// TODO: the actual CUDA porting work is work in progress...
//
//       For good performance we have to get rid of most local memory spills
//       TODO: make sure all loops have known trip counts at compile time
//             and are adequately unrolled.

#include <map>
#include <stdint.h>

#include "salsa_kernel.h"
#include "miner.h"

#include "keccak.h"

// define some error checking macros
#undef checkCudaErrors

#if WIN32
#define DELIMITER '\\'
#else
#define DELIMITER '/'
#endif
#define __FILENAME__ ( strrchr(__FILE__, DELIMITER) != NULL ? strrchr(__FILE__, DELIMITER)+1 : __FILE__ )

#define checkCudaErrors(x) \
{ \
    hipGetLastError(); \
    x; \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) \
        applog(LOG_ERR, "GPU #%d: hipError_t %d (%s) calling '%s' (%s line %d)\n", device_map[thr_id], err, hipGetErrorString(err), #x, __FILENAME__, __LINE__); \
}

// from salsa_kernel.cu
extern std::map<int, uint32_t *> context_idata[2];
extern std::map<int, uint32_t *> context_odata[2];
extern std::map<int, hipStream_t> context_streams[2];
extern std::map<int, uint32_t *> context_hash[2];

#define ROTL64(a,b) (((a) << (b)) | ((a) >> (64 - b)))

// CB
#define U32TO64_LE(p) \
    (((uint64_t)(*p)) | (((uint64_t)(*(p + 1))) << 32))

#define U64TO32_LE(p, v) \
    *p = (uint32_t)((v)); *(p+1) = (uint32_t)((v) >> 32);

static __device__ void mycpy64(uint32_t *d, const uint32_t *s) {
#pragma unroll 16
    for (int k=0; k < 16; ++k) d[k] = s[k];
}

static __device__ void mycpy56(uint32_t *d, const uint32_t *s) {
#pragma unroll 14
    for (int k=0; k < 14; ++k) d[k] = s[k];
}

static __device__ void mycpy32(uint32_t *d, const uint32_t *s) {
#pragma unroll 8
    for (int k=0; k < 8; ++k) d[k] = s[k];
}

static __device__ void mycpy8(uint32_t *d, const uint32_t *s) {
#pragma unroll 2
    for (int k=0; k < 2; ++k) d[k] = s[k];
}

static __device__ void mycpy4(uint32_t *d, const uint32_t *s) {
    *d = *s;
}

// ---------------------------- BEGIN keccak functions ------------------------------------

#define KECCAK_HASH "Keccak-512"

typedef struct keccak_hash_state_t {
    uint64_t state[25];                        // 25*2
    uint32_t buffer[72/4];                     // 72
} keccak_hash_state;

__device__ void statecopy0(keccak_hash_state *d, keccak_hash_state *s)
{
#pragma unroll 25
    for (int i=0; i < 25; ++i)
        d->state[i] = s->state[i];
}

__device__ void statecopy8(keccak_hash_state *d, keccak_hash_state *s)
{
#pragma unroll 25
    for (int i=0; i < 25; ++i)
        d->state[i] = s->state[i];
#pragma unroll 2
    for (int i=0; i < 2; ++i)
        d->buffer[i] = s->buffer[i];
}

static const uint64_t host_keccak_round_constants[24] = {
    0x0000000000000001ull, 0x0000000000008082ull,
    0x800000000000808aull, 0x8000000080008000ull,
    0x000000000000808bull, 0x0000000080000001ull,
    0x8000000080008081ull, 0x8000000000008009ull,
    0x000000000000008aull, 0x0000000000000088ull,
    0x0000000080008009ull, 0x000000008000000aull,
    0x000000008000808bull, 0x800000000000008bull,
    0x8000000000008089ull, 0x8000000000008003ull,
    0x8000000000008002ull, 0x8000000000000080ull,
    0x000000000000800aull, 0x800000008000000aull,
    0x8000000080008081ull, 0x8000000000008080ull,
    0x0000000080000001ull, 0x8000000080008008ull
};

__constant__ uint64_t c_keccak_round_constants[24];
__constant__ uint32_t pdata[20];

__device__ void
keccak_block(keccak_hash_state *S, const uint32_t *in) {
    size_t i;
    uint64_t *s = S->state, t[5], u[5], v, w;

    /* absorb input */
#pragma unroll 9
    for (i = 0; i < 72 / 8; i++, in += 2)
        s[i] ^= U32TO64_LE(in);
    
    for (i = 0; i < 24; i++) {
        /* theta: c = a[0,i] ^ a[1,i] ^ .. a[4,i] */
        t[0] = s[0] ^ s[5] ^ s[10] ^ s[15] ^ s[20];
        t[1] = s[1] ^ s[6] ^ s[11] ^ s[16] ^ s[21];
        t[2] = s[2] ^ s[7] ^ s[12] ^ s[17] ^ s[22];
        t[3] = s[3] ^ s[8] ^ s[13] ^ s[18] ^ s[23];
        t[4] = s[4] ^ s[9] ^ s[14] ^ s[19] ^ s[24];

        /* theta: d[i] = c[i+4] ^ rotl(c[i+1],1) */
        u[0] = t[4] ^ ROTL64(t[1], 1);
        u[1] = t[0] ^ ROTL64(t[2], 1);
        u[2] = t[1] ^ ROTL64(t[3], 1);
        u[3] = t[2] ^ ROTL64(t[4], 1);
        u[4] = t[3] ^ ROTL64(t[0], 1);

        /* theta: a[0,i], a[1,i], .. a[4,i] ^= d[i] */
        s[0] ^= u[0]; s[5] ^= u[0]; s[10] ^= u[0]; s[15] ^= u[0]; s[20] ^= u[0];
        s[1] ^= u[1]; s[6] ^= u[1]; s[11] ^= u[1]; s[16] ^= u[1]; s[21] ^= u[1];
        s[2] ^= u[2]; s[7] ^= u[2]; s[12] ^= u[2]; s[17] ^= u[2]; s[22] ^= u[2];
        s[3] ^= u[3]; s[8] ^= u[3]; s[13] ^= u[3]; s[18] ^= u[3]; s[23] ^= u[3];
        s[4] ^= u[4]; s[9] ^= u[4]; s[14] ^= u[4]; s[19] ^= u[4]; s[24] ^= u[4];

        /* rho pi: b[..] = rotl(a[..], ..) */
        v = s[ 1];
        s[ 1] = ROTL64(s[ 6], 44);
        s[ 6] = ROTL64(s[ 9], 20);
        s[ 9] = ROTL64(s[22], 61);
        s[22] = ROTL64(s[14], 39);
        s[14] = ROTL64(s[20], 18);
        s[20] = ROTL64(s[ 2], 62);
        s[ 2] = ROTL64(s[12], 43);
        s[12] = ROTL64(s[13], 25);
        s[13] = ROTL64(s[19],  8);
        s[19] = ROTL64(s[23], 56);
        s[23] = ROTL64(s[15], 41);
        s[15] = ROTL64(s[ 4], 27);
        s[ 4] = ROTL64(s[24], 14);
        s[24] = ROTL64(s[21],  2);
        s[21] = ROTL64(s[ 8], 55);
        s[ 8] = ROTL64(s[16], 45);
        s[16] = ROTL64(s[ 5], 36);
        s[ 5] = ROTL64(s[ 3], 28);
        s[ 3] = ROTL64(s[18], 21);
        s[18] = ROTL64(s[17], 15);
        s[17] = ROTL64(s[11], 10);
        s[11] = ROTL64(s[ 7],  6);
        s[ 7] = ROTL64(s[10],  3);
        s[10] = ROTL64(    v,  1);

        /* chi: a[i,j] ^= ~b[i,j+1] & b[i,j+2] */
        v = s[ 0]; w = s[ 1]; s[ 0] ^= (~w) & s[ 2]; s[ 1] ^= (~s[ 2]) & s[ 3]; s[ 2] ^= (~s[ 3]) & s[ 4]; s[ 3] ^= (~s[ 4]) & v; s[ 4] ^= (~v) & w;
        v = s[ 5]; w = s[ 6]; s[ 5] ^= (~w) & s[ 7]; s[ 6] ^= (~s[ 7]) & s[ 8]; s[ 7] ^= (~s[ 8]) & s[ 9]; s[ 8] ^= (~s[ 9]) & v; s[ 9] ^= (~v) & w;
        v = s[10]; w = s[11]; s[10] ^= (~w) & s[12]; s[11] ^= (~s[12]) & s[13]; s[12] ^= (~s[13]) & s[14]; s[13] ^= (~s[14]) & v; s[14] ^= (~v) & w;
        v = s[15]; w = s[16]; s[15] ^= (~w) & s[17]; s[16] ^= (~s[17]) & s[18]; s[17] ^= (~s[18]) & s[19]; s[18] ^= (~s[19]) & v; s[19] ^= (~v) & w;
        v = s[20]; w = s[21]; s[20] ^= (~w) & s[22]; s[21] ^= (~s[22]) & s[23]; s[22] ^= (~s[23]) & s[24]; s[23] ^= (~s[24]) & v; s[24] ^= (~v) & w;

        /* iota: a[0,0] ^= round constant */
        s[0] ^= c_keccak_round_constants[i];
    }
}

__device__ void
keccak_hash_init(keccak_hash_state *S) { 
#pragma unroll 25
    for (int i=0; i<25; ++i)
        S->state[i] = 0ULL;
}

// assuming there is no leftover data and exactly 72 bytes are incoming
// we can directly call into the block hashing function
__device__ void
keccak_hash_update72(keccak_hash_state *S, const uint32_t *in) {
    keccak_block(S, in);
}

__device__ void keccak_hash_update8(keccak_hash_state *S, const uint32_t *in) {
    mycpy8(S->buffer, in);
}

__device__ void keccak_hash_update4_8(keccak_hash_state *S, const uint32_t *in) {
    mycpy4(S->buffer+8/4, in);
}

__device__ void keccak_hash_update4_56(keccak_hash_state *S, const uint32_t *in) {
    mycpy4(S->buffer+56/4, in);
}

__device__ void keccak_hash_update56(keccak_hash_state *S, const uint32_t *in) {
    mycpy56(S->buffer, in);
}

__device__ void keccak_hash_update64(keccak_hash_state *S, const uint32_t *in) {
    mycpy64(S->buffer, in);
}

__device__ void
keccak_hash_finish8(keccak_hash_state *S, uint32_t *hash) {
    S->buffer[8/4] = 0x01;
#pragma unroll 15
    for (int i=8/4+1; i < 72/4; ++i) S->buffer[i] = 0;
    S->buffer[72/4 - 1] |= 0x80000000;
    keccak_block(S, (const uint32_t*)S->buffer);
#pragma unroll 8
    for (size_t i = 0; i < 64; i += 8) {
        U64TO32_LE((&hash[i/4]), S->state[i / 8]);
    }
}

__device__ void
keccak_hash_finish12(keccak_hash_state *S, uint32_t *hash) {
    S->buffer[12/4] = 0x01;
#pragma unroll 14
    for (int i=12/4+1; i < 72/4; ++i) S->buffer[i] = 0;
    S->buffer[72/4 - 1] |= 0x80000000;
    keccak_block(S, (const uint32_t*)S->buffer);
#pragma unroll 8
    for (size_t i = 0; i < 64; i += 8) {
        U64TO32_LE((&hash[i/4]), S->state[i / 8]);
    }
}

__device__ void
keccak_hash_finish60(keccak_hash_state *S, uint32_t *hash) {
    S->buffer[60/4] = 0x01;
#pragma unroll 2
    for (int i=60/4+1; i < 72/4; ++i) S->buffer[i] = 0;
    S->buffer[72/4 - 1] |= 0x80000000;
    keccak_block(S, (const uint32_t*)S->buffer);
#pragma unroll 8
    for (size_t i = 0; i < 64; i += 8) {
        U64TO32_LE((&hash[i/4]), S->state[i / 8]);
    }
}

__device__ void
keccak_hash_finish64(keccak_hash_state *S, uint32_t *hash) {
    S->buffer[64/4] = 0x01;
#pragma unroll 1
    for (int i=64/4+1; i < 72/4; ++i) S->buffer[i] = 0;
    S->buffer[72/4 - 1] |= 0x80000000;
    keccak_block(S, (const uint32_t*)S->buffer);
#pragma unroll 8
    for (size_t i = 0; i < 64; i += 8) {
        U64TO32_LE((&hash[i/4]), S->state[i / 8]);
    }
}

// ---------------------------- END keccak functions ------------------------------------

// ---------------------------- BEGIN PBKDF2 functions ------------------------------------

typedef struct pbkdf2_hmac_state_t {
    keccak_hash_state inner, outer;
} pbkdf2_hmac_state;


__device__ void
pbkdf2_hash(uint32_t *hash, const uint32_t *m) {
    keccak_hash_state st;
    keccak_hash_init(&st);
    keccak_hash_update72(&st, m);
    keccak_hash_update8(&st, m+72/4);
    keccak_hash_finish8(&st, hash);
}

/* hmac */
__device__ void
pbkdf2_hmac_init80(pbkdf2_hmac_state *st, const uint32_t *key) {
    uint32_t pad[72/4];
    size_t i;

    keccak_hash_init(&st->inner);
    keccak_hash_init(&st->outer);

#pragma unroll 18
    for (i = 0; i < 72/4; i++)
        pad[i] = 0;

    /* key > blocksize bytes, hash it */
    pbkdf2_hash(pad, key);

    /* inner = (key ^ 0x36) */
    /* h(inner || ...) */
#pragma unroll 18
    for (i = 0; i < 72/4; i++)
        pad[i] ^= 0x36363636;
    keccak_hash_update72(&st->inner, pad);

    /* outer = (key ^ 0x5c) */
    /* h(outer || ...) */
#pragma unroll 18
    for (i = 0; i < 72/4; i++)
        pad[i] ^= 0x6a6a6a6a;
    keccak_hash_update72(&st->outer, pad);
}

// assuming there is no leftover data and exactly 72 bytes are incoming
// we can directly call into the block hashing function
__device__ void
pbkdf2_hmac_update72(pbkdf2_hmac_state *st, const uint32_t *m) {
    /* h(inner || m...) */
    keccak_hash_update72(&st->inner, m);
}

__device__ void
pbkdf2_hmac_update8(pbkdf2_hmac_state *st, const uint32_t *m) {
    /* h(inner || m...) */
    keccak_hash_update8(&st->inner, m);
}

__device__ void
pbkdf2_hmac_update4_8(pbkdf2_hmac_state *st, const uint32_t *m) {
    /* h(inner || m...) */
    keccak_hash_update4_8(&st->inner, m);
}

__device__ void
pbkdf2_hmac_update4_56(pbkdf2_hmac_state *st, const uint32_t *m) {
    /* h(inner || m...) */
    keccak_hash_update4_56(&st->inner, m);
}

__device__ void
pbkdf2_hmac_update56(pbkdf2_hmac_state *st, const uint32_t *m) {
    /* h(inner || m...) */
    keccak_hash_update56(&st->inner, m);
}

__device__ void
pbkdf2_hmac_finish12(pbkdf2_hmac_state *st, uint32_t *mac) {
    /* h(inner || m) */
    uint32_t innerhash[16];
    keccak_hash_finish12(&st->inner, innerhash);

    /* h(outer || h(inner || m)) */
    keccak_hash_update64(&st->outer, innerhash);
    keccak_hash_finish64(&st->outer, mac);
}

__device__ void
pbkdf2_hmac_finish60(pbkdf2_hmac_state *st, uint32_t *mac) {
    /* h(inner || m) */
    uint32_t innerhash[16];
    keccak_hash_finish60(&st->inner, innerhash);

    /* h(outer || h(inner || m)) */
    keccak_hash_update64(&st->outer, innerhash);
    keccak_hash_finish64(&st->outer, mac);
}

__device__ void
pbkdf2_statecopy8(pbkdf2_hmac_state *d, pbkdf2_hmac_state *s) {
    statecopy8(&d->inner, &s->inner);
    statecopy0(&d->outer, &s->outer);
}

// ---------------------------- END PBKDF2 functions ------------------------------------

static __device__ uint32_t cuda_swab32(uint32_t x)
{
    return (((x << 24) & 0xff000000u) | ((x << 8) & 0x00ff0000u)
          | ((x >> 8) & 0x0000ff00u) | ((x >> 24) & 0x000000ffu));
}

__global__ __launch_bounds__(128) void cuda_pre_keccak512(uint32_t *g_idata, uint32_t nonce)
{
    nonce        +=       (blockIdx.x * blockDim.x) + threadIdx.x; 
    g_idata      += 32 * ((blockIdx.x * blockDim.x) + threadIdx.x);

    uint32_t data[20];

#pragma unroll 19
    for (int i=0; i <19; ++i)
        data[i] = cuda_swab32(pdata[i]);
    data[19] = cuda_swab32(nonce);

//    scrypt_pbkdf2_1((const uint8_t*)data, 80, (const uint8_t*)data, 80, (uint8_t*)g_idata, 128);

    pbkdf2_hmac_state hmac_pw, work;
    uint32_t ti[16];
    uint32_t be;
    
    /* hmac(password, ...) */
    pbkdf2_hmac_init80(&hmac_pw, data);

    /* hmac(password, salt...) */
    pbkdf2_hmac_update72(&hmac_pw, data);
    pbkdf2_hmac_update8(&hmac_pw, data+72/4);

    /* U1 = hmac(password, salt || be(i)) */
    be = cuda_swab32(1);
    pbkdf2_statecopy8(&work, &hmac_pw);
    pbkdf2_hmac_update4_8(&work, &be);
    pbkdf2_hmac_finish12(&work, ti);
    mycpy64(g_idata, ti);

    be = cuda_swab32(2);
    pbkdf2_statecopy8(&work, &hmac_pw);
    pbkdf2_hmac_update4_8(&work, &be);
    pbkdf2_hmac_finish12(&work, ti);
    mycpy64(g_idata+16, ti);
}


__global__ __launch_bounds__(128) void cuda_post_keccak512(uint32_t *g_odata, uint32_t *g_hash, uint32_t nonce)
{
    nonce        +=       (blockIdx.x * blockDim.x) + threadIdx.x; 
    g_odata      += 32 * ((blockIdx.x * blockDim.x) + threadIdx.x);
    g_hash       +=  8 * ((blockIdx.x * blockDim.x) + threadIdx.x);

    uint32_t data[20];

#pragma unroll 19
    for (int i=0; i <19; ++i)
        data[i] = cuda_swab32(pdata[i]);
    data[19] = cuda_swab32(nonce);

//    scrypt_pbkdf2_1((const uint8_t*)data, 80, (const uint8_t*)g_odata, 128, (uint8_t*)g_hash, 32);

    pbkdf2_hmac_state hmac_pw;
    uint32_t ti[16];
    uint32_t be;
    
    /* hmac(password, ...) */
    pbkdf2_hmac_init80(&hmac_pw, data);

    /* hmac(password, salt...) */
    pbkdf2_hmac_update72(&hmac_pw, g_odata);
    pbkdf2_hmac_update56(&hmac_pw, g_odata+72/4);

    /* U1 = hmac(password, salt || be(i)) */
    be = cuda_swab32(1);
    pbkdf2_hmac_update4_56(&hmac_pw, &be);
    pbkdf2_hmac_finish60(&hmac_pw, ti);
    mycpy32(g_hash, ti);
}

//
// callable host code to initialize constants and to call kernels
//

extern "C" void prepare_keccak512(int thr_id, const uint32_t host_pdata[20])
{
    static bool init[8] = {false, false, false, false, false, false, false, false};
    if (!init[thr_id])
    {
        hipMemcpyToSymbol(HIP_SYMBOL(c_keccak_round_constants), host_keccak_round_constants, sizeof(host_keccak_round_constants), 0, hipMemcpyHostToDevice);
        init[thr_id] = true;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(pdata), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}

extern "C" void pre_keccak512(int thr_id, int stream, uint32_t nonce, int throughput)
{
    dim3 block(128);
    dim3 grid((throughput+127)/128);

    cuda_pre_keccak512<<<grid, block, 0, context_streams[stream][thr_id]>>>(context_idata[stream][thr_id], nonce);
}

extern "C" void post_keccak512(int thr_id, int stream, uint32_t nonce, uint32_t hash[8], int throughput)
{
    dim3 block(128);
    dim3 grid((throughput+127)/128);

    cuda_post_keccak512<<<grid, block, 0, context_streams[stream][thr_id]>>>(context_odata[stream][thr_id], context_hash[stream][thr_id], nonce);

    unsigned int mem_size = throughput * sizeof(uint32_t) * 8;

    // copy device memory to host
    checkCudaErrors(hipMemcpyAsync(hash, context_hash[stream][thr_id], mem_size,
                    hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
}


//
// Maxcoin related Keccak implementation (Keccak256)
//

#include <stdint.h>

#include <map>
extern std::map<int, int> context_blocks;
extern std::map<int, int> context_wpb;
extern std::map<int, KernelInterface *> context_kernel;

__constant__ uint64_t ptarget64[4];

#define ROL(a, offset) ((((uint64_t)a) << ((offset) % 64)) ^ (((uint64_t)a) >> (64-((offset) % 64))))
#define ROL_mult8(a, offset) ROL(a, offset)

__constant__ uint64_t KeccakF_RoundConstants[24];

static uint64_t host_KeccakF_RoundConstants[24] = 
{
    (uint64_t)0x0000000000000001ULL,
    (uint64_t)0x0000000000008082ULL,
    (uint64_t)0x800000000000808aULL,
    (uint64_t)0x8000000080008000ULL,
    (uint64_t)0x000000000000808bULL,
    (uint64_t)0x0000000080000001ULL,
    (uint64_t)0x8000000080008081ULL,
    (uint64_t)0x8000000000008009ULL,
    (uint64_t)0x000000000000008aULL,
    (uint64_t)0x0000000000000088ULL,
    (uint64_t)0x0000000080008009ULL,
    (uint64_t)0x000000008000000aULL,
    (uint64_t)0x000000008000808bULL,
    (uint64_t)0x800000000000008bULL,
    (uint64_t)0x8000000000008089ULL,
    (uint64_t)0x8000000000008003ULL,
    (uint64_t)0x8000000000008002ULL,
    (uint64_t)0x8000000000000080ULL,
    (uint64_t)0x000000000000800aULL,
    (uint64_t)0x800000008000000aULL,
    (uint64_t)0x8000000080008081ULL,
    (uint64_t)0x8000000000008080ULL,
    (uint64_t)0x0000000080000001ULL,
    (uint64_t)0x8000000080008008ULL
};

__constant__ uint64_t pdata64[10];

__global__ void crypto_hash( uint64_t *g_out, uint32_t nonce, uint32_t *g_good, bool validate )
{
    uint64_t Aba, Abe, Abi, Abo, Abu;
    uint64_t Aga, Age, Agi, Ago, Agu;
    uint64_t Aka, Ake, Aki, Ako, Aku;
    uint64_t Ama, Ame, Ami, Amo, Amu;
    uint64_t Asa, Ase, Asi, Aso, Asu;
    uint64_t BCa, BCe, BCi, BCo, BCu;
    uint64_t Da, De, Di, Do, Du;
    uint64_t Eba, Ebe, Ebi, Ebo, Ebu;
    uint64_t Ega, Ege, Egi, Ego, Egu;
    uint64_t Eka, Eke, Eki, Eko, Eku;
    uint64_t Ema, Eme, Emi, Emo, Emu;
    uint64_t Esa, Ese, Esi, Eso, Esu;

    //copyFromState(A, state)
    Aba = pdata64[0];
    Abe = pdata64[1];
    Abi = pdata64[2];
    Abo = pdata64[3];
    Abu = pdata64[4];
    Aga = pdata64[5];
    Age = pdata64[6];
    Agi = pdata64[7];
    Ago = pdata64[8];
    Agu = (pdata64[9] & 0x00000000FFFFFFFFULL) | (((uint64_t)cuda_swab32(nonce + ((blockIdx.x * blockDim.x) + threadIdx.x))) << 32);
    Aka = 0x0000000000000001ULL;
    Ake = 0;
    Aki = 0;
    Ako = 0;
    Aku = 0;
    Ama = 0;
    Ame = 0x8000000000000000ULL;
    Ami = 0;
    Amo = 0;
    Amu = 0;
    Asa = 0;
    Ase = 0;
    Asi = 0;
    Aso = 0;
    Asu = 0;

#pragma unroll 12
    for( int laneCount = 0; laneCount < 24; laneCount += 2 )
    {
        //    prepareTheta
        BCa = Aba^Aga^Aka^Ama^Asa;
        BCe = Abe^Age^Ake^Ame^Ase;
        BCi = Abi^Agi^Aki^Ami^Asi;
        BCo = Abo^Ago^Ako^Amo^Aso;
        BCu = Abu^Agu^Aku^Amu^Asu;

        //thetaRhoPiChiIotaPrepareTheta(round  , A, E)
        Da = BCu^ROL(BCe, 1);
        De = BCa^ROL(BCi, 1);
        Di = BCe^ROL(BCo, 1);
        Do = BCi^ROL(BCu, 1);
        Du = BCo^ROL(BCa, 1);

        Aba ^= Da;
        BCa = Aba;
        Age ^= De;
        BCe = ROL(Age, 44);
        Aki ^= Di;
        BCi = ROL(Aki, 43);
        Amo ^= Do;
        BCo = ROL(Amo, 21);
        Asu ^= Du;
        BCu = ROL(Asu, 14);
        Eba =   BCa ^((~BCe)&  BCi );
        Eba ^= (uint64_t)KeccakF_RoundConstants[laneCount];
        Ebe =   BCe ^((~BCi)&  BCo );
        Ebi =   BCi ^((~BCo)&  BCu );
        Ebo =   BCo ^((~BCu)&  BCa );
        Ebu =   BCu ^((~BCa)&  BCe );

        Abo ^= Do;
        BCa = ROL(Abo, 28);
        Agu ^= Du;
        BCe = ROL(Agu, 20);
        Aka ^= Da;
        BCi = ROL(Aka,  3);
        Ame ^= De;
        BCo = ROL(Ame, 45);
        Asi ^= Di;
        BCu = ROL(Asi, 61);
        Ega =   BCa ^((~BCe)&  BCi );
        Ege =   BCe ^((~BCi)&  BCo );
        Egi =   BCi ^((~BCo)&  BCu );
        Ego =   BCo ^((~BCu)&  BCa );
        Egu =   BCu ^((~BCa)&  BCe );

        Abe ^= De;
        BCa = ROL(Abe,  1);
        Agi ^= Di;
        BCe = ROL(Agi,  6);
        Ako ^= Do;
        BCi = ROL(Ako, 25);
        Amu ^= Du;
        BCo = ROL_mult8(Amu,  8);
        Asa ^= Da;
        BCu = ROL(Asa, 18);
        Eka =   BCa ^((~BCe)&  BCi );
        Eke =   BCe ^((~BCi)&  BCo );
        Eki =   BCi ^((~BCo)&  BCu );
        Eko =   BCo ^((~BCu)&  BCa );
        Eku =   BCu ^((~BCa)&  BCe );

        Abu ^= Du;
        BCa = ROL(Abu, 27);
        Aga ^= Da;
        BCe = ROL(Aga, 36);
        Ake ^= De;
        BCi = ROL(Ake, 10);
        Ami ^= Di;
        BCo = ROL(Ami, 15);
        Aso ^= Do;
        BCu = ROL_mult8(Aso, 56);
        Ema =   BCa ^((~BCe)&  BCi );
        Eme =   BCe ^((~BCi)&  BCo );
        Emi =   BCi ^((~BCo)&  BCu );
        Emo =   BCo ^((~BCu)&  BCa );
        Emu =   BCu ^((~BCa)&  BCe );

        Abi ^= Di;
        BCa = ROL(Abi, 62);
        Ago ^= Do;
        BCe = ROL(Ago, 55);
        Aku ^= Du;
        BCi = ROL(Aku, 39);
        Ama ^= Da;
        BCo = ROL(Ama, 41);
        Ase ^= De;
        BCu = ROL(Ase,  2);
        Esa =   BCa ^((~BCe)&  BCi );
        Ese =   BCe ^((~BCi)&  BCo );
        Esi =   BCi ^((~BCo)&  BCu );
        Eso =   BCo ^((~BCu)&  BCa );
        Esu =   BCu ^((~BCa)&  BCe );

        //    prepareTheta
        BCa = Eba^Ega^Eka^Ema^Esa;
        BCe = Ebe^Ege^Eke^Eme^Ese;
        BCi = Ebi^Egi^Eki^Emi^Esi;
        BCo = Ebo^Ego^Eko^Emo^Eso;
        BCu = Ebu^Egu^Eku^Emu^Esu;

        //thetaRhoPiChiIotaPrepareTheta(round+1, E, A)
        Da = BCu^ROL(BCe, 1);
        De = BCa^ROL(BCi, 1);
        Di = BCe^ROL(BCo, 1);
        Do = BCi^ROL(BCu, 1);
        Du = BCo^ROL(BCa, 1);

        Eba ^= Da;
        BCa = Eba;
        Ege ^= De;
        BCe = ROL(Ege, 44);
        Eki ^= Di;
        BCi = ROL(Eki, 43);
        Emo ^= Do;
        BCo = ROL(Emo, 21);
        Esu ^= Du;
        BCu = ROL(Esu, 14);
        Aba =   BCa ^((~BCe)&  BCi );
        Aba ^= (uint64_t)KeccakF_RoundConstants[laneCount+1];
        Abe =   BCe ^((~BCi)&  BCo );
        Abi =   BCi ^((~BCo)&  BCu );
        Abo =   BCo ^((~BCu)&  BCa );
        Abu =   BCu ^((~BCa)&  BCe );

        Ebo ^= Do;
        BCa = ROL(Ebo, 28);
        Egu ^= Du;
        BCe = ROL(Egu, 20);
        Eka ^= Da;
        BCi = ROL(Eka, 3);
        Eme ^= De;
        BCo = ROL(Eme, 45);
        Esi ^= Di;
        BCu = ROL(Esi, 61);
        Aga =   BCa ^((~BCe)&  BCi );
        Age =   BCe ^((~BCi)&  BCo );
        Agi =   BCi ^((~BCo)&  BCu );
        Ago =   BCo ^((~BCu)&  BCa );
        Agu =   BCu ^((~BCa)&  BCe );

        Ebe ^= De;
        BCa = ROL(Ebe, 1);
        Egi ^= Di;
        BCe = ROL(Egi, 6);
        Eko ^= Do;
        BCi = ROL(Eko, 25);
        Emu ^= Du;
        BCo = ROL_mult8(Emu, 8);
        Esa ^= Da;
        BCu = ROL(Esa, 18);
        Aka =   BCa ^((~BCe)&  BCi );
        Ake =   BCe ^((~BCi)&  BCo );
        Aki =   BCi ^((~BCo)&  BCu );
        Ako =   BCo ^((~BCu)&  BCa );
        Aku =   BCu ^((~BCa)&  BCe );

        Ebu ^= Du;
        BCa = ROL(Ebu, 27);
        Ega ^= Da;
        BCe = ROL(Ega, 36);
        Eke ^= De;
        BCi = ROL(Eke, 10);
        Emi ^= Di;
        BCo = ROL(Emi, 15);
        Eso ^= Do;
        BCu = ROL_mult8(Eso, 56);
        Ama =   BCa ^((~BCe)&  BCi );
        Ame =   BCe ^((~BCi)&  BCo );
        Ami =   BCi ^((~BCo)&  BCu );
        Amo =   BCo ^((~BCu)&  BCa );
        Amu =   BCu ^((~BCa)&  BCe );

        Ebi ^= Di;
        BCa = ROL(Ebi, 62);
        Ego ^= Do;
        BCe = ROL(Ego, 55);
        Eku ^= Du;
        BCi = ROL(Eku, 39);
        Ema ^= Da;
        BCo = ROL(Ema, 41);
        Ese ^= De;
        BCu = ROL(Ese, 2);
        Asa =   BCa ^((~BCe)&  BCi );
        Ase =   BCe ^((~BCi)&  BCo );
        Asi =   BCi ^((~BCo)&  BCu );
        Aso =   BCo ^((~BCu)&  BCa );
        Asu =   BCu ^((~BCa)&  BCe );
    }

    if (validate) {
        g_out += 4 * ((blockIdx.x * blockDim.x) + threadIdx.x);
        g_out[3] = Abo;
        g_out[2] = Abi;
        g_out[1] = Abe;
        g_out[0] = Aba;
    }
    
    // the likelyhood of meeting the hashing target is so low, that we're not guarding this
    // with atomic writes, locks or similar...
    uint64_t *g_good64 = (uint64_t*)g_good;
    if (Abo <=  ptarget64[3]) {
        if (Abo < g_good64[3]) {
            g_good64[3] = Abo;
            g_good64[2] = Abi;
            g_good64[1] = Abe;
            g_good64[0] = Aba;
            g_good[8] = nonce + ((blockIdx.x * blockDim.x) + threadIdx.x);
        }
    }
}

static std::map<int, uint32_t *> context_good[2];

extern "C" void default_prepare_keccak256(int thr_id, const uint32_t host_pdata[20], const uint32_t host_ptarget[8])
{
    static bool init[8] = {false, false, false, false, false, false, false, false};
    if (!init[thr_id])
    {
        hipMemcpyToSymbol(HIP_SYMBOL(KeccakF_RoundConstants), host_KeccakF_RoundConstants, sizeof(host_KeccakF_RoundConstants), 0, hipMemcpyHostToDevice);

        // allocate pinned host memory for good hashes
        uint32_t *tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[0][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, 9*sizeof(uint32_t))); context_good[1][thr_id] = tmp;

        init[thr_id] = true;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(pdata64), host_pdata, 20*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(ptarget64), host_ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice);
}

extern "C" uint32_t default_do_keccak256(int thr_id, int stream, uint32_t *hash, uint32_t nonce, int throughput, bool do_d2h)
{
    uint32_t result = 0xffffffff;
  
    unsigned int GRID_BLOCKS = context_blocks[thr_id];
    unsigned int WARPS_PER_BLOCK = context_wpb[thr_id];
    unsigned int THREADS_PER_WU = context_kernel[thr_id]->threads_per_wu();

    // setup execution parameters
    dim3  grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
    dim3  threads(THREADS_PER_WU*WU_PER_BLOCK, 1, 1);

    checkCudaErrors(hipMemsetAsync(context_good[stream][thr_id], 0xff, 9 * sizeof(uint32_t), context_streams[stream][thr_id]));

    crypto_hash<<<grid, threads, 0, context_streams[stream][thr_id]>>>((uint64_t*)context_hash[stream][thr_id], nonce, context_good[stream][thr_id], do_d2h);

    // copy hashes from device memory to host (ALL hashes, lots of data...)
    if (do_d2h) {
        size_t mem_size = throughput * sizeof(uint32_t) * 8;
        checkCudaErrors(hipMemcpyAsync(hash, context_hash[stream][thr_id], mem_size,
                        hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
    }
    else {
        // asynchronous copy of winning nonce (just 4 bytes...)
        checkCudaErrors(hipMemcpyAsync(hash, context_good[stream][thr_id]+8, sizeof(uint32_t),
                        hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
    }

    return result;
}
