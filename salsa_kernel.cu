#include "hip/hip_runtime.h"

//
// Contains the autotuning logic and some utility functions.
// Note that all CUDA kernels have been moved to other .cu files
//
// NOTE: compile this .cu module for compute_10,sm_10 with --maxrregcount=64
//

#ifdef WIN32
#include <windows.h>
#endif
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <ctype.h>

#include <map>
#include <algorithm>

#include <hip/hip_runtime.h>

#include "salsa_kernel.h"

#include "titan_kernel.h"
#include "fermi_kernel.h"
#include "test_kernel.h"
#include "nv_kernel.h"
#include "nv_kernel2.h"
#include "kepler_kernel.h"

#include "miner.h"

#if WIN32
#ifdef _WIN64
#define _64BIT 1
#endif
#else
#if __x86_64__
#define _64BIT 1
#endif
#endif

#if _64BIT
#define MAXMEM 0x300000000ULL  // 12 GB (the largest Kepler)
#else
#define MAXMEM  0xFFFFFFFFULL  // nearly 4 GB (32 bit limitations)
#endif

// require CUDA 5.5 driver API
#define DMAJ 5
#define DMIN 5

// define some error checking macros
#undef checkCudaErrors

#if WIN32
#define DELIMITER '/'
#else
#define DELIMITER '/'
#endif
#define __FILENAME__ ( strrchr(__FILE__, DELIMITER) != NULL ? strrchr(__FILE__, DELIMITER)+1 : __FILE__ )

#define checkCudaErrors(x) \
{ \
    hipGetLastError(); \
    x; \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) \
        applog(LOG_ERR, "GPU #%d: hipError_t %d (%s) calling '%s' (%s line %d)\n", device_map[thr_id], err, hipGetErrorString(err), #x, __FILENAME__, __LINE__); \
}

// some globals containing pointers to device memory (for chunked allocation)
// [MAX_DEVICES] indexes up to MAX_DEVICES threads (0...MAX_DEVICES-1)
int       MAXWARPS[MAX_DEVICES];
uint32_t* h_V[MAX_DEVICES][TOTAL_WARP_LIMIT*64];          // NOTE: the *64 prevents buffer overflow for --keccak
uint32_t  h_V_extra[MAX_DEVICES][TOTAL_WARP_LIMIT*64];    //       with really large kernel launch configurations

extern "C" int cuda_num_devices()
{
    int version;
    hipError_t err = hipDriverGetVersion(&version);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "FATAL: Unable to query CUDA driver version! Is an nVidia driver installed?");
        return -1;
    }

    int maj = version / 1000, min = version % 100; // same as in deviceQuery sample
    if (maj < DMAJ || (maj == DMAJ && min < DMIN))
    {
        applog(LOG_ERR, "FATAL: Driver does not support CUDA %d.%d API! Update your nVidia driver!", DMAJ, DMIN);
        return -1;
    }

    int GPU_N;
    err = hipGetDeviceCount(&GPU_N);
    if (err != hipSuccess)
    {
        applog(LOG_ERR, "FATAL: Unable to query number of CUDA devices! Is an nVidia driver installed?");
        return -1;
    }
    return GPU_N;
}

static bool substringsearch(const char *haystack, const char *needle, int &match)
{
    int hlen = strlen(haystack);
    int nlen = strlen(needle);
    for (int i=0; i < hlen; ++i)
    {
        if (haystack[i] == ' ') continue;
        int j=0, x = 0;
        while(j < nlen)
        {
            if (haystack[i+x] == ' ') {++x; continue;}
            if (needle[j] == ' ') {++j; continue;}
            if (needle[j] == '#') return ++match == needle[j+1]-'0';
            if (tolower(haystack[i+x]) != tolower(needle[j])) break;
            ++j; ++x;
        }
        if (j == nlen) return true;
    }
    return false;
}

extern "C" int cuda_finddevice(char *name)
{
    int num = cuda_num_devices();
    int match = 0;
    for (int i=0; i < num; ++i)
    {
        hipDeviceProp_t props;
        if (hipGetDeviceProperties(&props, i) == hipSuccess)
            if (substringsearch(props.name, name, match)) return i;
    }
    return -1;
}

KernelInterface *Best_Kernel_Heuristics(hipDeviceProp_t *props)
{
    KernelInterface *kernel = NULL;
    if (opt_algo == ALGO_SCRYPT || (opt_algo == ALGO_SCRYPT_JANE && N <= 8192) || opt_algo == ALGO_KECCAK || opt_algo == ALGO_BLAKE)
    {
        // high register count kernels (scrypt, low N-factor scrypt-jane)
        if (props->major > 3 || (props->major == 3 && props->minor >= 5))
            kernel = new NV2Kernel(); // we don't want this for Keccak though
        else if (props->major == 3 && props->minor == 0)
            kernel = new NVKernel();
        else if (props->major == 2 || props->major == 1)
            kernel = new FermiKernel();
    }
    else
    {
       // low register count kernels (high N-factor scrypt-jane)
       if (props->major > 3 || (props->major == 3 && props->minor >= 5))
            kernel = new TitanKernel();
        else if (props->major == 3 && props->minor == 0)
            kernel = new KeplerKernel();
        else if (props->major == 2 || props->major == 1)
            kernel = new TestKernel();
    }
    return kernel;
}


bool validate_config(char *config, int &b, int &w, KernelInterface **kernel = NULL, hipDeviceProp_t *props = NULL)
{
    bool success = false;
    char kernelid = ' ';
    if (config != NULL)
    {
        if (config[0] == 'T' || config[0] == 'K' || config[0] == 'F' || config[0] == 'L' ||
            config[0] == 't' || config[0] == 'k' || config[0] == 'f' ||
            config[0] == 'Z' || config[0] == 'Y' || config[0] == 'X') {
            kernelid = config[0];
            config++;
        }

        if (config[0] >= '0' && config[0] <= '9')
            if (sscanf(config, "%dx%d", &b, &w) == 2)
                success = true;

        if (success && kernel != NULL)
        {
            switch (kernelid)
            {
                case 'T': case 'Z': *kernel = new NV2Kernel(); break;
                case 't':           *kernel = new TitanKernel(); break;
                case 'K': case 'Y': *kernel = new NVKernel(); break;
                case 'k':           *kernel = new KeplerKernel(); break;
                case 'F': case 'L': *kernel = new FermiKernel(); break;
                case 'f': case 'X': *kernel = new TestKernel(); break;
                case ' ': // choose based on device architecture
                    *kernel = Best_Kernel_Heuristics(props);
                break;
            }
        }
    }
    return success;
}

std::map<int, int> context_blocks;
std::map<int, int> context_wpb;
std::map<int, bool> context_concurrent;
std::map<int, KernelInterface *> context_kernel;
std::map<int, uint32_t *> context_idata[2];
std::map<int, uint32_t *> context_odata[2];
std::map<int, hipStream_t> context_streams[2];
std::map<int, uint32_t *> context_X[2];
std::map<int, uint32_t *> context_H[2];
std::map<int, hipEvent_t> context_serialize[2];

// for SHA256 hashing on GPU
std::map<int, uint32_t *> context_tstate[2];
std::map<int, uint32_t *> context_ostate[2];
std::map<int, uint32_t *> context_hash[2];

int find_optimal_blockcount(int thr_id, KernelInterface* &kernel, bool &concurrent, int &wpb);

extern "C" void cuda_shutdown(int thr_id)
{
    hipDeviceSynchronize();
    hipDeviceReset();
    hipDeviceReset();
}

extern "C" int cuda_throughput(int thr_id)
{
    int GRID_BLOCKS, WARPS_PER_BLOCK;
    if (context_blocks.find(thr_id) == context_blocks.end())
    {
#if 0
        hipCtx_t ctx;
        hipCtxCreate( &ctx, hipDeviceScheduleYield, device_map[thr_id] );
        hipCtxSetCurrent(ctx);
#else
        checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleYield));
        checkCudaErrors(hipSetDevice(device_map[thr_id]));
        checkCudaErrors(hipFree(0));
#endif

        KernelInterface *kernel;
        bool concurrent; 
        GRID_BLOCKS = find_optimal_blockcount(thr_id, kernel, concurrent, WARPS_PER_BLOCK);

        if(GRID_BLOCKS == 0)
            return 0;

        unsigned int THREADS_PER_WU = kernel->threads_per_wu();
        unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * 32;
        unsigned int state_size = WU_PER_LAUNCH * sizeof(uint32_t) * 8;

        // allocate device memory for scrypt_core inputs and outputs
        uint32_t *tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_idata[0][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_idata[1][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_odata[0][thr_id] = tmp;
        checkCudaErrors(hipMalloc((void **) &tmp, mem_size)); context_odata[1][thr_id] = tmp;

        // allocate pinned host memory for scrypt hashes
        checkCudaErrors(hipHostAlloc((void **) &tmp, state_size, hipHostMallocDefault)); context_H[0][thr_id] = tmp;
        checkCudaErrors(hipHostAlloc((void **) &tmp, state_size, hipHostMallocDefault)); context_H[1][thr_id] = tmp;

        if (opt_algo == ALGO_SCRYPT)
        {
            if (parallel < 2)
            {
                // allocate pinned host memory for scrypt_core input/output
                checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[0][thr_id] = tmp;
                checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[1][thr_id] = tmp;
            }
            else
            {
                // allocate tstate, ostate, scrypt hash device memory
                checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_tstate[0][thr_id] = tmp;
                checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_tstate[1][thr_id] = tmp;
                checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_ostate[0][thr_id] = tmp;
                checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_ostate[1][thr_id] = tmp;
                checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[0][thr_id] = tmp;
                checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[1][thr_id] = tmp;
            }
        }
        else if (opt_algo == ALGO_SCRYPT_JANE)
        {
            // allocate pinned host memory for scrypt_core input/output
            checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[0][thr_id] = tmp;
            checkCudaErrors(hipHostAlloc((void **) &tmp, mem_size, hipHostMallocDefault)); context_X[1][thr_id] = tmp;

            checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[0][thr_id] = tmp;
            checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[1][thr_id] = tmp;
        }
        else if (opt_algo == ALGO_KECCAK || opt_algo == ALGO_BLAKE)
        {
            checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[0][thr_id] = tmp;
            checkCudaErrors(hipMalloc((void **) &tmp, state_size)); context_hash[1][thr_id] = tmp;
        }

        // create two CUDA streams
        hipStream_t tmp2;
        checkCudaErrors( hipStreamCreate(&tmp2) ); context_streams[0][thr_id] = tmp2;
        checkCudaErrors( hipStreamCreate(&tmp2) ); context_streams[1][thr_id] = tmp2;

        // events used to serialize the kernel launches (we don't want any overlapping of kernels)
        hipEvent_t tmp4;
        checkCudaErrors(hipEventCreateWithFlags(&tmp4, hipEventDisableTiming)); context_serialize[0][thr_id] = tmp4;
        checkCudaErrors(hipEventCreateWithFlags(&tmp4, hipEventDisableTiming)); context_serialize[1][thr_id] = tmp4;
        checkCudaErrors(hipEventRecord(context_serialize[1][thr_id]));

        context_kernel[thr_id] = kernel;
        context_concurrent[thr_id] = concurrent;
        context_blocks[thr_id] = GRID_BLOCKS;
        context_wpb[thr_id] = WARPS_PER_BLOCK;
    }

    GRID_BLOCKS = context_blocks[thr_id];
    WARPS_PER_BLOCK = context_wpb[thr_id];
    unsigned int THREADS_PER_WU = context_kernel[thr_id]->threads_per_wu();
    return WU_PER_LAUNCH;
}

// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x10, 8 }, // Tesla Generation (SM 1.0) G80 class
        { 0x11, 8 }, // Tesla Generation (SM 1.1) G8x class
        { 0x12, 8 }, // Tesla Generation (SM 1.2) G9x class
        { 0x13, 8 }, // Tesla Generation (SM 1.3) GT200 class
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }

    // If we don't find the values, we default use the previous one to run properly
//    printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[7].Cores);
    return nGpuArchCoresPerSM[7].Cores;
}

#ifdef WIN32
#include <windows.h>
static int console_width()
{
    CONSOLE_SCREEN_BUFFER_INFO csbi;
    GetConsoleScreenBufferInfo(GetStdHandle(STD_OUTPUT_HANDLE), &csbi);
    return csbi.srWindow.Right - csbi.srWindow.Left + 1;
}
#else
int console_width()
{
    return 999;
}
#endif

int find_optimal_blockcount(int thr_id, KernelInterface* &kernel, bool &concurrent, int &WARPS_PER_BLOCK)
{
    int cw = console_width();
    int optimal_blocks = 0;

    hipDeviceProp_t props;
    checkCudaErrors(hipGetDeviceProperties(&props, device_map[thr_id]));
    concurrent = (props.concurrentKernels > 0);

    device_name[thr_id] = strdup(props.name);
    applog(LOG_INFO, "GPU #%d: %s with compute capability %d.%d", device_map[thr_id], props.name, props.major, props.minor);

    WARPS_PER_BLOCK = -1;

    // if not specified, use interactive mode for devices that have the watchdog timer enabled
    if (device_interactive[thr_id] == -1)
        device_interactive[thr_id] = props.kernelExecTimeoutEnabled;

    // turn off texture cache if not otherwise specified
    if (device_texturecache[thr_id] == -1)
        device_texturecache[thr_id] = 0;

    // if not otherwise specified or required, turn single memory allocations off as they reduce
    // the amount of memory that we can allocate on Windows Vista, 7 and 8 (WDDM driver model issue)
    if (device_singlememory[thr_id] == -1) device_singlememory[thr_id] = 0;

    // figure out which kernel implementation to use
    if (!validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK, &kernel, &props)) {
        kernel = NULL;
        if (device_config[thr_id] != NULL) {
                 if (device_config[thr_id][0] == 'T' || device_config[thr_id][0] == 'Z')
                kernel = new NV2Kernel();
            else if (device_config[thr_id][0] == 't')
                kernel = new TitanKernel();
            else if (device_config[thr_id][0] == 'K' || device_config[thr_id][0] == 'Y')
                kernel = new NVKernel();
            else if (device_config[thr_id][0] == 'k')
                kernel = new KeplerKernel();
            else if (device_config[thr_id][0] == 'F' || device_config[thr_id][0] == 'L')
                kernel = new FermiKernel();
            else if (device_config[thr_id][0] == 'f' || device_config[thr_id][0] == 'X')
                kernel = new TestKernel();
        }
        if (kernel == NULL) kernel = Best_Kernel_Heuristics(&props);
    }

    if (kernel->get_major_version() > props.major || kernel->get_major_version() == props.major && kernel->get_minor_version() > props.minor)
    {
        applog(LOG_ERR, "GPU #%d: FATAL: the '%c' kernel requires %d.%d capability!", device_map[thr_id], kernel->get_identifier(), kernel->get_major_version(), kernel->get_minor_version());
        return 0;
    }

    // set whatever cache configuration and shared memory bank mode the kernel prefers
    checkCudaErrors(hipDeviceSetCacheConfig(kernel->cache_config()));
    checkCudaErrors(hipDeviceSetSharedMemConfig(kernel->shared_mem_config()));

    // some kernels (e.g. Titan) do not support the texture cache
    if (kernel->no_textures() && device_texturecache[thr_id]) {
        applog(LOG_WARNING, "GPU #%d: the '%c' kernel ignores the texture cache argument", device_map[thr_id], kernel->get_identifier());
        device_texturecache[thr_id] = 0;
    }

    // Texture caching only works with single memory allocation
    if (device_texturecache[thr_id]) device_singlememory[thr_id] = 1;

    if (kernel->single_memory() && !device_singlememory[thr_id]) {
        applog(LOG_WARNING, "GPU #%d: the '%c' kernel requires single memory allocation", device_map[thr_id], kernel->get_identifier());
        device_singlememory[thr_id] = 1;
    }

    if (device_lookup_gap[thr_id] == 0) device_lookup_gap[thr_id] = 1;
    if (!kernel->support_lookup_gap() && device_lookup_gap[thr_id] > 1)
    {
        applog(LOG_WARNING, "GPU #%d: the '%c' kernel does not support a lookup gap", device_map[thr_id], kernel->get_identifier());
        device_lookup_gap[thr_id] = 1;
    }

    applog(LOG_INFO, "GPU #%d: interactive: %d, tex-cache: %d%c, single-alloc: %d", device_map[thr_id],
           (device_interactive[thr_id]  != 0) ? 1 : 0,
           (device_texturecache[thr_id] != 0) ? device_texturecache[thr_id] : 0, (device_texturecache[thr_id] != 0) ? 'D' : ' ',
           (device_singlememory[thr_id] != 0) ? 1 : 0 );

    // number of threads collaborating on one work unit (hash)
    unsigned int THREADS_PER_WU = kernel->threads_per_wu();
    unsigned int LOOKUP_GAP = device_lookup_gap[thr_id];
    unsigned int BACKOFF = device_backoff[thr_id];
    applog(LOG_INFO, "GPU #%d: %d hashes / %.1f MB per warp.", device_map[thr_id], WU_PER_WARP, ((double)SCRATCH * WU_PER_WARP * sizeof(uint32_t) / (1024 * 1024)));

    // compute highest MAXWARPS numbers for kernels allowing hipBindTexture to succeed
    int MW_1D_4 = 134217728 / (SCRATCH * WU_PER_WARP / 4); // for uint4_t textures
    int MW_1D_2 = 134217728 / (SCRATCH * WU_PER_WARP / 2); // for uint2_t textures
    int MW_1D = kernel->get_texel_width() == 2 ? MW_1D_2 : MW_1D_4;

    uint32_t *d_V = NULL;
    if (device_singlememory[thr_id])
    {
        // if no launch config was specified, we simply
        // allocate the single largest memory chunk on the device that we can get
        if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK)) {
            MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
        }
        else {
            // compute no. of warps to allocate the largest number producing a single memory block
            // PROBLEM: one some devices, ALL allocations will fail if the first one failed. This sucks.
            size_t MEM_LIMIT = (size_t)min((unsigned long long)MAXMEM, (unsigned long long)props.totalGlobalMem);
            int warpmax = (int)min((unsigned long long)TOTAL_WARP_LIMIT, (unsigned long long)MEM_LIMIT / (SCRATCH * WU_PER_WARP * sizeof(uint32_t)));

            // run a bisection algorithm for memory allocation (way more reliable than the previous approach)
            int best = 0;
            int warp = (warpmax+1)/2;
            int interval = (warpmax+1)/2;
            while (interval > 0)
            {
                hipGetLastError(); // clear the error state
                hipMalloc((void **)&d_V, (size_t)SCRATCH * WU_PER_WARP * warp * sizeof(uint32_t));
                if (hipGetLastError() == hipSuccess) {
                    checkCudaErrors(hipFree(d_V)); d_V = NULL;
                    if (warp > best) best = warp;
                    if (warp == warpmax) break;
                    interval = (interval+1)/2;
                    warp += interval;
                    if (warp > warpmax) warp = warpmax;
                }
                else
                {
                    interval = interval/2;
                    warp -= interval;
                    if (warp < 1) warp = 1;
                }
            }
            // back off a bit from the largest possible allocation size
            MAXWARPS[thr_id] = ((100-BACKOFF)*best+50)/100;
        }

        // now allocate a buffer for determined MAXWARPS setting
        hipGetLastError(); // clear the error state
        hipMalloc((void **)&d_V, (size_t)SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
        if (hipGetLastError() == hipSuccess) {
            for (int i=0; i < MAXWARPS[thr_id]; ++i)
                h_V[thr_id][i] = d_V + SCRATCH * WU_PER_WARP * i;

            if (device_texturecache[thr_id] == 1)
            {
                if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
                {
                    if ( optimal_blocks * WARPS_PER_BLOCK > MW_1D ) {
                        applog(LOG_ERR, "GPU #%d: '%s' exceeds limits for 1D cache. Using 2D cache instead.", device_map[thr_id], device_config[thr_id]);
                        device_texturecache[thr_id] = 2;
                    }
                }
                // bind linear memory to a 1D texture reference
                if (kernel->get_texel_width() == 2)
                    kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * std::min(MAXWARPS[thr_id],MW_1D_2) * sizeof(uint32_t));
                else
                    kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * std::min(MAXWARPS[thr_id],MW_1D_4) * sizeof(uint32_t));
            }
            else if (device_texturecache[thr_id] == 2)
            {
                // bind pitch linear memory to a 2D texture reference
                if (kernel->get_texel_width() == 2)
                    kernel->bindtexture_2D(d_V, SCRATCH/2, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
                else
                    kernel->bindtexture_2D(d_V, SCRATCH/4, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
            }
        }
        else
        {
            applog(LOG_ERR, "GPU #%d: FATAL: Launch config '%s' requires too much memory!", device_map[thr_id], device_config[thr_id]);
            return 0;
        }
    }
    else
    {
        if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
            MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
        else
            MAXWARPS[thr_id] = TOTAL_WARP_LIMIT;

        // chunked memory allocation up to device limits
        int warp;
        for (warp = 0; warp < MAXWARPS[thr_id]; ++warp) {
            // work around partition camping problems by adding a random start address offset to each allocation
            h_V_extra[thr_id][warp] = (props.major == 1) ? (16 * (rand()%(16384/16))) : 0;
            hipGetLastError(); // clear the error state
            hipMalloc((void **) &h_V[thr_id][warp], (SCRATCH * WU_PER_WARP + h_V_extra[thr_id][warp])*sizeof(uint32_t));
            if (hipGetLastError() == hipSuccess) h_V[thr_id][warp] += h_V_extra[thr_id][warp];
            else {
                h_V_extra[thr_id][warp] = 0;

                // back off by several warp allocations to have some breathing room
                int remove = (BACKOFF*warp+50)/100;
                for (int i=0; warp > 0 && i < remove; ++i) {
                    warp--;
                    checkCudaErrors(hipFree(h_V[thr_id][warp]-h_V_extra[thr_id][warp]));
                    h_V[thr_id][warp] = NULL; h_V_extra[thr_id][warp] = 0;
                }

                break;
            }
        }
        MAXWARPS[thr_id] = warp;
    }
    if (opt_algo == ALGO_SCRYPT || opt_algo == ALGO_SCRYPT_JANE) kernel->set_scratchbuf_constants(MAXWARPS[thr_id], h_V[thr_id]);

    if (validate_config(device_config[thr_id], optimal_blocks, WARPS_PER_BLOCK))
    {
        if (optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
        {
            applog(LOG_ERR, "GPU #%d: FATAL: Given launch config '%s' requires too much memory.", device_map[thr_id], device_config[thr_id]);
            return 0;
        }

        if (WARPS_PER_BLOCK > kernel->max_warps_per_block())
        {
            applog(LOG_ERR, "GPU #%d: FATAL: Given launch config '%s' exceeds warp limit for '%c' kernel.", device_map[thr_id], device_config[thr_id], kernel->get_identifier());
            return 0;
        }
    }
    else
    {
        if (device_config[thr_id] != NULL && strcasecmp("auto", device_config[thr_id]))
            applog(LOG_WARNING, "GPU #%d: Given launch config '%s' does not validate.", device_map[thr_id], device_config[thr_id]);

        if (autotune)
        {
            applog(LOG_INFO, "GPU #%d: Performing auto-tuning (Patience...)", device_map[thr_id]);

            // allocate device memory
            uint32_t *d_idata = NULL, *d_odata = NULL;
            if (opt_algo == ALGO_SCRYPT || opt_algo == ALGO_SCRYPT_JANE) {
                unsigned int mem_size = MAXWARPS[thr_id] * WU_PER_WARP * sizeof(uint32_t) * 32;
                checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
                checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

                // pre-initialize some device memory
                uint32_t *h_idata = (uint32_t*)malloc(mem_size);
                for (unsigned int i=0; i < mem_size/sizeof(uint32_t); ++i) h_idata[i] = i*2654435761UL; // knuth's method
                checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));
                free(h_idata);
            } else if (opt_algo == ALGO_KECCAK) {
                uint32_t pdata[20] = {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20};
                uint32_t ptarget[8] = {0,0,0,0,0,0,0,0};
                kernel->prepare_keccak256(thr_id, pdata, ptarget);
            } else if (opt_algo == ALGO_BLAKE) {
                uint32_t pdata[20] = {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20};
                uint32_t ptarget[8] = {0,0,0,0,0,0,0,0};
                kernel->prepare_blake256(thr_id, pdata, ptarget);
            }

            double best_hash_sec = 0.0;
            int best_wpb = 0;

            // auto-tuning loop
            {
                // we want to have enough total warps for half the multiprocessors at least
                // compute highest MAXWARPS number that we can support based on texture cache mode
                int MINTW = props.multiProcessorCount / 2;
                int MAXTW = (device_texturecache[thr_id] == 1) ? std::min(MAXWARPS[thr_id],MW_1D) : MAXWARPS[thr_id];

                // we want to have blocks for half the multiprocessors at least
                int MINB = props.multiProcessorCount / 2;
                int MAXB = MAXTW;

                double tmin = 0.05;
                if (opt_algo == ALGO_KECCAK || opt_algo == ALGO_BLAKE) tmin = 0.01;

                applog(LOG_INFO, "GPU #%d: maximum total warps (BxW): %d", device_map[thr_id], MAXTW);

                for (int GRID_BLOCKS = MINB; !abort_flag && GRID_BLOCKS <= MAXB; ++GRID_BLOCKS)
                {
                    double Hash[32+1] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0 };
                    for (WARPS_PER_BLOCK = 1; !abort_flag && WARPS_PER_BLOCK <= kernel->max_warps_per_block(); ++WARPS_PER_BLOCK)
                    {
                        double hash_sec = 0;
                        if (GRID_BLOCKS * WARPS_PER_BLOCK >= MINTW &&
                            GRID_BLOCKS * WARPS_PER_BLOCK <= MAXTW)
                        {
                            // setup execution parameters
                            dim3  grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
                            dim3  threads(THREADS_PER_WU*WU_PER_BLOCK, 1, 1);

                            struct timeval tv_start, tv_end;
                            double tdelta = 0;

                            checkCudaErrors(hipDeviceSynchronize());
                            gettimeofday(&tv_start, NULL);
                            int repeat = 0;
                            do  // average several measurements for better exactness
                            {
                                if (opt_algo == ALGO_SCRYPT || opt_algo == ALGO_SCRYPT_JANE)
                                    kernel->run_kernel(grid, threads, WARPS_PER_BLOCK, thr_id, NULL, d_idata, d_odata, N, LOOKUP_GAP, device_interactive[thr_id], true, device_texturecache[thr_id]);
                                else if (opt_algo == ALGO_KECCAK)
                                    kernel->do_keccak256(grid, threads, thr_id, 0, NULL, rand(), WU_PER_LAUNCH, false);
                                else if (opt_algo == ALGO_BLAKE)
                                    kernel->do_blake256(grid, threads, thr_id, 0, NULL, rand(), WU_PER_LAUNCH, false);
                                if(hipDeviceSynchronize() != hipSuccess)
                                    break;
                                ++repeat;
                                gettimeofday(&tv_end, NULL);
                                // for a better result averaging, measure for at least 50ms (10ms for Keccak)
                            } while ((tdelta=(1e-6 * (tv_end.tv_usec-tv_start.tv_usec) + (tv_end.tv_sec-tv_start.tv_sec))) < tmin);
                            if (hipGetLastError() != hipSuccess) continue;

                            tdelta /= repeat; // BUGFIX: this averaging over multiple measurements was missing

                            // for scrypt: in interactive mode only find launch configs where kernel launch times are short enough
                            // TODO: instead we could reduce the batchsize parameter to meet the launch time requirement.
                            if (opt_algo == ALGO_SCRYPT && device_interactive[thr_id] && GRID_BLOCKS > 2*props.multiProcessorCount && tdelta > 1.0/30)
                                if (WARPS_PER_BLOCK == 1) goto skip; else goto skip2;

                            hash_sec = (double)WU_PER_LAUNCH / tdelta;
                            Hash[WARPS_PER_BLOCK] = hash_sec;
                            if (hash_sec > best_hash_sec) {
                                optimal_blocks = GRID_BLOCKS;
                                best_hash_sec = hash_sec;
                                best_wpb = WARPS_PER_BLOCK;
                            }
                        }
                    }
skip2:              ;
                    if (opt_debug) {
                        if (GRID_BLOCKS == MINB) {
                            char line[512] = "    ";
                            for (int i=1; i<=kernel->max_warps_per_block(); ++i) {
                                char tmp[16]; sprintf(tmp, i < 10 ? "   x%-2d" : "  x%-2d ", i);
                                strcat(line, tmp);
                                if (cw == 80 && (i % 8 == 0 && i != kernel->max_warps_per_block()))
                                    strcat(line, "\n                          ");
                            }
                            applog(LOG_DEBUG, line);
                        }

                        char kMGT = ' '; bool flag;
                        for (int j=0; j < 4; ++j) {
                            flag=false; for (int i=1; i<=kernel->max_warps_per_block(); flag|=Hash[i] >= 1000, i++);
                            if (flag)   for (int i=1; i<=kernel->max_warps_per_block(); Hash[i] /= 1000, i++);
                            else break;
                                 if (kMGT == ' ') kMGT = 'k';
                            else if (kMGT == 'k') kMGT = 'M';
                            else if (kMGT == 'M') kMGT = 'G';
                            else if (kMGT == 'G') kMGT = 'T';
                        }
                        char *format = "%5.4f%c";
                        flag = false; for (int i=1; i<=kernel->max_warps_per_block(); flag|=Hash[i] >= 1, i++); if (flag) format = "%5.3f%c";
                        flag = false; for (int i=1; i<=kernel->max_warps_per_block(); flag|=Hash[i] >= 10, i++); if (flag) format = "%5.2f%c";
                        flag = false; for (int i=1; i<=kernel->max_warps_per_block(); flag|=Hash[i] >= 100, i++); if (flag) format = "%5.1f%c";

                        char line[512]; sprintf(line, "%3d:", GRID_BLOCKS);
                        for (int i=1; i<=kernel->max_warps_per_block(); ++i) {
                            char tmp[16];
                            if (Hash[i]>0)
                                sprintf(tmp, format, Hash[i], (i<kernel->max_warps_per_block())?'|':' ');
                            else
                                sprintf(tmp, "     %c", (i<kernel->max_warps_per_block())?'|':' ');
                            strcat(line, tmp);
                            if (cw == 80 && (i % 8 == 0 && i != kernel->max_warps_per_block()))
                                strcat(line, "\n                          ");
                        }
                        int n = strlen(line)-1; line[n++] = '|'; line[n++] = ' '; line[n++] = kMGT; line[n++] = '\0';
                        strcat(line, "H/s");
                        applog(LOG_DEBUG, line);
                    }
                }
skip:           ;
            }

            if (opt_algo == ALGO_SCRYPT || opt_algo == ALGO_SCRYPT_JANE) {
                checkCudaErrors(hipFree(d_odata));
                checkCudaErrors(hipFree(d_idata));
            }

            WARPS_PER_BLOCK = best_wpb;
            applog(LOG_INFO, "GPU #%d: %7.2f hash/s with configuration %c%dx%d", device_map[thr_id], best_hash_sec, kernel->get_identifier(), optimal_blocks, WARPS_PER_BLOCK);
        }
        else
        {
            // Heuristics for finding a good kernel launch configuration

            // base the initial block estimate on the number of multiprocessors
            int device_cores = props.multiProcessorCount * _ConvertSMVer2Cores(props.major, props.minor);

            // defaults, in case nothing else is chosen below
            optimal_blocks = 4 * device_cores / WU_PER_WARP;
            WARPS_PER_BLOCK = 2;

            // Based on compute capability, pick a known good block x warp configuration.
            if (props.major == 3)
            {
                if (props.minor == 0) // GK104, GK106, GK107
                {
                    if (MAXWARPS[thr_id] > (int)(optimal_blocks * 1.7261905) * 2)
                    {
                        // this results in 290x2 configuration on GTX 660Ti (3GB)
                        // but it requires 3GB memory on the card!
                        optimal_blocks = (int)(optimal_blocks * 1.7261905);
                        WARPS_PER_BLOCK = 2;
                    }
                    else
                    {
                        // this results in 148x2 configuration on GTX 660Ti (2GB)
                        optimal_blocks = (int)(optimal_blocks * 0.8809524);
                        WARPS_PER_BLOCK = 2;
                    }
                }
                else if (props.minor == 5) // GK110 (Tesla K20X, K20, GeForce GTX TITAN)
                {
                    // TODO: what to do with Titan and Tesla K20(X)?
                    // for now, do the same as for GTX 660Ti (2GB)
                    optimal_blocks = (int)(optimal_blocks * 0.8809524);
                    WARPS_PER_BLOCK = 2;
                }
            }
            // 1st generation Fermi (compute 2.0) GF100, GF110
            else if (props.major == 2 && props.minor == 0)
            {
                // this results in a 60x4 configuration on GTX 570
                optimal_blocks = 4 * device_cores / WU_PER_WARP;
                WARPS_PER_BLOCK = 4;
            }
            // 2nd generation Fermi (compute 2.1) GF104,106,108,114,116
            else if (props.major == 2 && props.minor == 1)
            {
                // this results in a 56x2 configuration on GTX 460
                optimal_blocks = props.multiProcessorCount * 8;
                WARPS_PER_BLOCK = 2;
            }
            // G80, G92, GT2xx
            else if (props.major == 1)
            {
                if (props.minor == 0)  // G80
                {
                    // TODO: anyone knowing good settings for G80?
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 4;
                }
                else if (props.minor == 1)  // G92
                {
                    // e.g. my 9600M works best at 4x4
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 4;
                }
                else if (props.minor == 2)  // GT218, GT216, GT215
                {
                    // TODO: anyone knowing good settings for Compute 1.2?
                    // for now I assume performance is identical to compute 1.3
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 3;
                }
                if (props.minor == 3)  // GT200
                {
                    // my GTX 260 works best at S27x3
                    optimal_blocks = props.multiProcessorCount;
                    WARPS_PER_BLOCK = 3;
                }
            }

            // in case we run out of memory with the automatically chosen configuration,
            // first back off with WARPS_PER_BLOCK, then reduce optimal_blocks.
            if (WARPS_PER_BLOCK==3 && optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
                WARPS_PER_BLOCK = 2;
            while (optimal_blocks > 0 && optimal_blocks * WARPS_PER_BLOCK > MAXWARPS[thr_id])
                optimal_blocks--;
        }
    }

    applog(LOG_INFO, "GPU #%d: using launch configuration %c%dx%d", device_map[thr_id], kernel->get_identifier(), optimal_blocks, WARPS_PER_BLOCK);

    if (device_singlememory[thr_id])
    {
        if (MAXWARPS[thr_id] != optimal_blocks * WARPS_PER_BLOCK)
        {
            MAXWARPS[thr_id] = optimal_blocks * WARPS_PER_BLOCK;
            if (device_texturecache[thr_id] == 1)
                kernel->unbindtexture_1D();
            else if (device_texturecache[thr_id] == 2)
                kernel->unbindtexture_2D();
            checkCudaErrors(hipFree(d_V)); d_V = NULL;

            hipGetLastError(); // clear the error state
            hipMalloc((void **)&d_V, (size_t)SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
            if (hipGetLastError() == hipSuccess) {
                for (int i=0; i < MAXWARPS[thr_id]; ++i)
                    h_V[thr_id][i] = d_V + SCRATCH * WU_PER_WARP * i;

                if (device_texturecache[thr_id] == 1)
                {
                    // bind linear memory to a 1D texture reference
                    if (kernel->get_texel_width() == 2)
                        kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
                    else
                        kernel->bindtexture_1D(d_V, SCRATCH * WU_PER_WARP * MAXWARPS[thr_id] * sizeof(uint32_t));
                }
                else if (device_texturecache[thr_id] == 2)
                {
                    // bind pitch linear memory to a 2D texture reference
                    if (kernel->get_texel_width() == 2)
                        kernel->bindtexture_2D(d_V, SCRATCH/2, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
                    else
                        kernel->bindtexture_2D(d_V, SCRATCH/4, WU_PER_WARP * MAXWARPS[thr_id], SCRATCH*sizeof(uint32_t));
                }

                // update pointers to scratch buffer in constant memory after reallocation
                if (opt_algo == ALGO_SCRYPT || opt_algo == ALGO_SCRYPT_JANE) kernel->set_scratchbuf_constants(MAXWARPS[thr_id], h_V[thr_id]);
            }
            else
            {
                applog(LOG_ERR, "GPU #%d: Unable to allocate enough memory for launch config '%s'.", device_map[thr_id], device_config[thr_id]);
            }
        }
    }
    else
    {
        // back off unnecessary memory allocations to have some breathing room
        while (MAXWARPS[thr_id] > 0 && MAXWARPS[thr_id] > optimal_blocks * WARPS_PER_BLOCK) {
            (MAXWARPS[thr_id])--;
            checkCudaErrors(hipFree(h_V[thr_id][MAXWARPS[thr_id]]-h_V_extra[thr_id][MAXWARPS[thr_id]]));
            h_V[thr_id][MAXWARPS[thr_id]] = NULL; h_V_extra[thr_id][MAXWARPS[thr_id]] = 0;
        }
    }

    return optimal_blocks;
}

extern "C" void cuda_scrypt_HtoD(int thr_id, uint32_t *X, int stream)
{
    unsigned int GRID_BLOCKS = context_blocks[thr_id];
    unsigned int WARPS_PER_BLOCK = context_wpb[thr_id];
    unsigned int THREADS_PER_WU = context_kernel[thr_id]->threads_per_wu();
    unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * 32;

    // copy host memory to device
    hipMemcpyAsync(context_idata[stream][thr_id], X, mem_size, hipMemcpyHostToDevice, context_streams[stream][thr_id]);
}

extern "C" void cuda_scrypt_serialize(int thr_id, int stream)
{
    // if the device can concurrently execute multiple kernels, then we must
    // wait for the serialization event recorded by the other stream
    //if (context_concurrent[thr_id] || device_interactive[thr_id])
        hipStreamWaitEvent(context_streams[stream][thr_id], context_serialize[(stream+1)&1][thr_id], 0);
}

extern "C" void cuda_scrypt_done(int thr_id, int stream)
{
    // record the serialization event in the current stream
    hipEventRecord(context_serialize[stream][thr_id], context_streams[stream][thr_id]);
}

extern "C" void cuda_scrypt_flush(int thr_id, int stream)
{
    // flush the work queue (required for WDDM drivers)
    hipStreamQuery(context_streams[stream][thr_id]);
}

extern "C" void cuda_scrypt_core(int thr_id, int stream, unsigned int N)
{
    unsigned int GRID_BLOCKS = context_blocks[thr_id];
    unsigned int WARPS_PER_BLOCK = context_wpb[thr_id];
    unsigned int THREADS_PER_WU = context_kernel[thr_id]->threads_per_wu();
    unsigned int LOOKUP_GAP = device_lookup_gap[thr_id];

    // setup execution parameters
    dim3  grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
    dim3  threads(THREADS_PER_WU*WU_PER_BLOCK, 1, 1);

    context_kernel[thr_id]->run_kernel(grid, threads, WARPS_PER_BLOCK, thr_id, context_streams[stream][thr_id], context_idata[stream][thr_id], context_odata[stream][thr_id], N, LOOKUP_GAP, device_interactive[thr_id], opt_benchmark, device_texturecache[thr_id]);
}

extern "C" bool cuda_prepare_keccak256(int thr_id, const uint32_t host_pdata[20], const uint32_t ptarget[8])
{
    return context_kernel[thr_id]->prepare_keccak256(thr_id, host_pdata, ptarget);
}

extern "C" void cuda_do_keccak256(int thr_id, int stream, uint32_t *hash, uint32_t nonce, int throughput, bool do_d2h)
{
    unsigned int GRID_BLOCKS = context_blocks[thr_id];
    unsigned int WARPS_PER_BLOCK = context_wpb[thr_id];
    unsigned int THREADS_PER_WU = context_kernel[thr_id]->threads_per_wu();

    // setup execution parameters
    dim3  grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
    dim3  threads(THREADS_PER_WU*WU_PER_BLOCK, 1, 1);

    context_kernel[thr_id]->do_keccak256(grid, threads, thr_id, stream, hash, nonce, throughput, do_d2h);
}

extern "C" bool cuda_prepare_blake256(int thr_id, const uint32_t host_pdata[20], const uint32_t ptarget[8])
{
    return context_kernel[thr_id]->prepare_blake256(thr_id, host_pdata, ptarget);
}

extern "C" void cuda_do_blake256(int thr_id, int stream, uint32_t *hash, uint32_t nonce, int throughput, bool do_d2h)
{
    unsigned int GRID_BLOCKS = context_blocks[thr_id];
    unsigned int WARPS_PER_BLOCK = context_wpb[thr_id];
    unsigned int THREADS_PER_WU = context_kernel[thr_id]->threads_per_wu();

    // setup execution parameters
    dim3  grid(WU_PER_LAUNCH/WU_PER_BLOCK, 1, 1);
    dim3  threads(THREADS_PER_WU*WU_PER_BLOCK, 1, 1);

    context_kernel[thr_id]->do_blake256(grid, threads, thr_id, stream, hash, nonce, throughput, do_d2h);
}

extern "C" void cuda_scrypt_DtoH(int thr_id, uint32_t *X, int stream, bool postSHA)
{
    unsigned int GRID_BLOCKS = context_blocks[thr_id];
    unsigned int WARPS_PER_BLOCK = context_wpb[thr_id];
    unsigned int THREADS_PER_WU = context_kernel[thr_id]->threads_per_wu();
    unsigned int mem_size = WU_PER_LAUNCH * sizeof(uint32_t) * (postSHA ? 8 : 32);

    // copy result from device to host (asynchronously)
    checkCudaErrors(hipMemcpyAsync(X, postSHA ? context_hash[stream][thr_id] : context_odata[stream][thr_id], mem_size, hipMemcpyDeviceToHost, context_streams[stream][thr_id]));
}

extern "C" bool cuda_scrypt_sync(int thr_id, int stream)
{
    hipError_t err;
    
    if(device_interactive[thr_id] && !opt_benchmark)
    {
        // For devices that also do desktop rendering or compositing, we want to free up some time slots.
        // That requires making a pause in work submission when there is no active task on the GPU,
        // and Device Synchronize ensures that.

        // this call was replaced by the loop below to workaround the high CPU usage issue
        //err = hipDeviceSynchronize();

        while((err = hipStreamQuery(context_streams[0][thr_id])) == hipErrorNotReady || 
              (err == hipSuccess && (err = hipStreamQuery(context_streams[1][thr_id])) == hipErrorNotReady)) 
            usleep(1000);

        usleep(1000);
    }
    else
    {
        // this call was replaced by the loop below to workaround the high CPU usage issue
        //err = hipStreamSynchronize(context_streams[stream][thr_id]);

        while((err = hipStreamQuery(context_streams[stream][thr_id])) == hipErrorNotReady)
            usleep(1000);
    }

    if(err != hipSuccess)
    {
        applog(LOG_ERR, "GPU #%d: CUDA error `%s` while executing the kernel.", device_map[thr_id], hipGetErrorString(err));
        return false;
    }

    return true;
}

extern "C" uint32_t* cuda_transferbuffer(int thr_id, int stream)
{
    return context_X[stream][thr_id];
}

extern "C" uint32_t* cuda_hashbuffer(int thr_id, int stream)
{
    return context_H[stream][thr_id];
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set on the CPU
//! @param idata      input data as provided to device
//! @param reference  reference data, computed but preallocated
//! @param V          scrypt scratchpad
////////////////////////////////////////////////////////////////////////////////
static void xor_salsa8(uint32_t * const B, const uint32_t * const C);

extern "C" void
computeGold(uint32_t *idata, uint32_t *reference, uint32_t *V)
{
    uint32_t X[32];
    unsigned int i; int j,k;

    for (k = 0; k < 32; k++)
        X[k] = idata[k];
    
    for (i = 0; i < N; i++) {
        memcpy(&V[i * 32], X, 128);
        xor_salsa8(&X[0], &X[16]);
        xor_salsa8(&X[16], &X[0]);
    }
    for (i = 0; i < N; i++) {
        j = 32 * (X[16] % N);
        for (k = 0; k < 32; k++)
            X[k] ^= V[j + k];
        xor_salsa8(&X[0], &X[16]);
        xor_salsa8(&X[16], &X[0]);
    }
    for (k = 0; k < 32; k++)
        reference[k] = X[k];
}

#define ROTL(a, b) (((a) << (b)) | ((a) >> (32 - (b))))

static void xor_salsa8(uint32_t * const B, const uint32_t * const C)
{
    uint32_t x0 = (B[ 0] ^= C[ 0]), x1 = (B[ 1] ^= C[ 1]), x2 = (B[ 2] ^= C[ 2]), x3 = (B[ 3] ^= C[ 3]);
    uint32_t x4 = (B[ 4] ^= C[ 4]), x5 = (B[ 5] ^= C[ 5]), x6 = (B[ 6] ^= C[ 6]), x7 = (B[ 7] ^= C[ 7]);
    uint32_t x8 = (B[ 8] ^= C[ 8]), x9 = (B[ 9] ^= C[ 9]), xa = (B[10] ^= C[10]), xb = (B[11] ^= C[11]);
    uint32_t xc = (B[12] ^= C[12]), xd = (B[13] ^= C[13]), xe = (B[14] ^= C[14]), xf = (B[15] ^= C[15]);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);

    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);

    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    /* Operate on columns. */
    x4 ^= ROTL(x0 + xc,  7);  x9 ^= ROTL(x5 + x1,  7); xe ^= ROTL(xa + x6,  7);  x3 ^= ROTL(xf + xb,  7);
    x8 ^= ROTL(x4 + x0,  9);  xd ^= ROTL(x9 + x5,  9); x2 ^= ROTL(xe + xa,  9);  x7 ^= ROTL(x3 + xf,  9);
    xc ^= ROTL(x8 + x4, 13);  x1 ^= ROTL(xd + x9, 13); x6 ^= ROTL(x2 + xe, 13);  xb ^= ROTL(x7 + x3, 13);
    x0 ^= ROTL(xc + x8, 18);  x5 ^= ROTL(x1 + xd, 18); xa ^= ROTL(x6 + x2, 18);  xf ^= ROTL(xb + x7, 18);
        
    /* Operate on rows. */
    x1 ^= ROTL(x0 + x3,  7);  x6 ^= ROTL(x5 + x4,  7); xb ^= ROTL(xa + x9,  7);  xc ^= ROTL(xf + xe,  7);
    x2 ^= ROTL(x1 + x0,  9);  x7 ^= ROTL(x6 + x5,  9); x8 ^= ROTL(xb + xa,  9);  xd ^= ROTL(xc + xf,  9);
    x3 ^= ROTL(x2 + x1, 13);  x4 ^= ROTL(x7 + x6, 13); x9 ^= ROTL(x8 + xb, 13);  xe ^= ROTL(xd + xc, 13);
    x0 ^= ROTL(x3 + x2, 18);  x5 ^= ROTL(x4 + x7, 18); xa ^= ROTL(x9 + x8, 18);  xf ^= ROTL(xe + xd, 18);

    B[ 0] += x0; B[ 1] += x1; B[ 2] += x2; B[ 3] += x3; B[ 4] += x4; B[ 5] += x5; B[ 6] += x6; B[ 7] += x7;
    B[ 8] += x8; B[ 9] += x9; B[10] += xa; B[11] += xb; B[12] += xc; B[13] += xd; B[14] += xe; B[15] += xf;
}

