/*
 * A trivial little dlopen()-based wrapper library for the
 * NVIDIA NVML library, to allow runtime discovery of NVML on an
 * arbitrary system.  This is all very hackish and simple-minded, but
 * it serves my immediate needs in the short term until NVIDIA provides
 * a static NVML wrapper library themselves, hopefully in
 * CUDA 6.5 or maybe sometime shortly after.
 *
 * This trivial code is made available under the "new" 3-clause BSD license,
 * and/or any of the GPL licenses you prefer.
 * Feel free to use the code and modify as you see fit.
 *
 * John E. Stone - john.stone@gmail.com
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include "wrapnvml.h"
#include "hip/hip_runtime.h"

/*
 * Wrappers to emulate dlopen() on other systems like Windows
 */
#if defined(_MSC_VER) || defined(_WIN32) || defined(_WIN64)
#include <windows.h>
static void *wrap_dlopen(const char *filename) {
  return (void *)LoadLibrary(filename);
}
static void *wrap_dlsym(void *h, const char *sym) {
  return (void *)GetProcAddress((HINSTANCE)h, sym);
}
static int wrap_dlclose(void *h) {
  /* FreeLibrary returns nonzero on success */
  return (!FreeLibrary((HINSTANCE)h));
}
#else
/* assume we can use dlopen itself... */
#include <dlfcn.h>
static void *wrap_dlopen(const char *filename) {
  return dlopen(filename, RTLD_NOW);
}
static void *wrap_dlsym(void *h, const char *sym) {
  return dlsym(h, sym);
}
static int wrap_dlclose(void *h) {
  return dlclose(h);
}
#endif

#if defined(__cplusplus)
extern "C" {
#endif

wrap_nvml_handle * wrap_nvml_create() {
  int i=0;
  wrap_nvml_handle *nvmlh = NULL;

  /* 
   * We use hard-coded library installation locations for the time being...
   * No idea where or if libnvidia-ml.so is installed on MacOS X, a 
   * deep scouring of the filesystem on one of the Mac CUDA build boxes
   * I used turned up nothing, so for now it's not going to work on OSX.
   */
#if defined(_WIN64)
  /* 64-bit Windows */
#define  libnvidia_ml "%PROGRAMFILES%/NVIDIA Corporation/NVSMI/nvml.dll"
#elif defined(_WIN32) || defined(_MSC_VER)
  /* 32-bit Windows */
#define  libnvidia_ml "%PROGRAMFILES%/NVIDIA Corporation/NVSMI/nvml.dll"
#elif defined(__linux) && (defined(__i386__) || defined(__ARM_ARCH_7A__))
  /* 32-bit linux assumed */
#define  libnvidia_ml "/usr/lib32/libnvidia-ml.so"
#elif defined(__linux)
  /* 64-bit linux assumed */
#define  libnvidia_ml "/usr/lib/libnvidia-ml.so"
#elif defined(__APPLE__)
#define  libnvidia_ml ""
#else
#error "Unrecognized platform: need NVML DLL path for this platform..."
#endif

#if WIN32
  char tmp[512];
  ExpandEnvironmentStringsA(libnvidia_ml, tmp, sizeof(tmp)); 
#else
  char tmp[512] = libnvidia_ml;
#endif

  void *nvml_dll = wrap_dlopen(tmp);
  if (nvml_dll == NULL)
    return NULL;

  nvmlh = (wrap_nvml_handle *) calloc(1, sizeof(wrap_nvml_handle));

  nvmlh->nvml_dll = nvml_dll;  

  nvmlh->nvmlInit = (wrap_nvmlReturn_t (*)(void)) 
    wrap_dlsym(nvmlh->nvml_dll, "nvmlInit");
  nvmlh->nvmlDeviceGetCount = (wrap_nvmlReturn_t (*)(int *)) 
    wrap_dlsym(nvmlh->nvml_dll, "nvmlDeviceGetCount_v2");
  nvmlh->nvmlDeviceGetHandleByIndex = (wrap_nvmlReturn_t (*)(int, wrap_nvmlDevice_t *)) 
    wrap_dlsym(nvmlh->nvml_dll, "nvmlDeviceGetHandleByIndex_v2");
  nvmlh->nvmlDeviceGetPciInfo = (wrap_nvmlReturn_t (*)(wrap_nvmlDevice_t, wrap_nvmlPciInfo_t *)) 
    wrap_dlsym(nvmlh->nvml_dll, "nvmlDeviceGetPciInfo");
  nvmlh->nvmlDeviceGetName = (wrap_nvmlReturn_t (*)(wrap_nvmlDevice_t, char *, int))
    wrap_dlsym(nvmlh->nvml_dll, "nvmlDeviceGetName");
  nvmlh->nvmlDeviceGetTemperature = (wrap_nvmlReturn_t (*)(wrap_nvmlDevice_t, int, unsigned int *))
    wrap_dlsym(nvmlh->nvml_dll, "nvmlDeviceGetTemperature");
  nvmlh->nvmlDeviceGetFanSpeed = (wrap_nvmlReturn_t (*)(wrap_nvmlDevice_t, unsigned int *))
    wrap_dlsym(nvmlh->nvml_dll, "nvmlDeviceGetFanSpeed");
  nvmlh->nvmlDeviceGetPowerUsage = (wrap_nvmlReturn_t (*)(wrap_nvmlDevice_t, unsigned int *))
    wrap_dlsym(nvmlh->nvml_dll, "nvmlDeviceGetPowerUsage");
  nvmlh->nvmlShutdown = (wrap_nvmlReturn_t (*)()) 
    wrap_dlsym(nvmlh->nvml_dll, "nvmlShutdown");

  if (nvmlh->nvmlInit == NULL || 
      nvmlh->nvmlShutdown == NULL ||
      nvmlh->nvmlDeviceGetCount == NULL ||
      nvmlh->nvmlDeviceGetHandleByIndex == NULL || 
      nvmlh->nvmlDeviceGetPciInfo == NULL ||
      nvmlh->nvmlDeviceGetName == NULL ||
      nvmlh->nvmlDeviceGetTemperature == NULL ||
      nvmlh->nvmlDeviceGetFanSpeed == NULL ||
      nvmlh->nvmlDeviceGetPowerUsage == NULL
      ) {
#if 0
    printf("Failed to obtain all required NVML function pointers\n");
#endif
    wrap_dlclose(nvmlh->nvml_dll);
    free(nvmlh);
    return NULL;
  }

  nvmlh->nvmlInit();
  nvmlh->nvmlDeviceGetCount(&nvmlh->nvml_gpucount);

  /* Query CUDA device count, in case it doesn't agree with NVML, since  */
  /* CUDA will only report GPUs with compute capability greater than 1.0 */ 
  if (hipGetDeviceCount(&nvmlh->cuda_gpucount) != hipSuccess) {
#if 0
    printf("Failed to query CUDA device count!\n");
#endif
    wrap_dlclose(nvmlh->nvml_dll);
    free(nvmlh);
    return NULL;
  }

  nvmlh->devs = (wrap_nvmlDevice_t *) calloc(nvmlh->nvml_gpucount, sizeof(wrap_nvmlDevice_t));
  nvmlh->nvml_pci_domain_id = (unsigned int*) calloc(nvmlh->nvml_gpucount, sizeof(unsigned int));
  nvmlh->nvml_pci_bus_id = (unsigned int*) calloc(nvmlh->nvml_gpucount, sizeof(unsigned int));
  nvmlh->nvml_pci_device_id = (unsigned int*) calloc(nvmlh->nvml_gpucount, sizeof(unsigned int));
  nvmlh->nvml_cuda_device_id = (int*) calloc(nvmlh->nvml_gpucount, sizeof(int));
  nvmlh->cuda_nvml_device_id = (int*) calloc(nvmlh->cuda_gpucount, sizeof(int));

  /* Obtain GPU device handles we're going to need repeatedly... */
  for (i=0; i<nvmlh->nvml_gpucount; i++) {
    nvmlh->nvmlDeviceGetHandleByIndex(i, &nvmlh->devs[i]);
  } 

  /* Query PCI info for each NVML device, and build table for mapping of */
  /* CUDA device IDs to NVML device IDs and vice versa                   */
  for (i=0; i<nvmlh->nvml_gpucount; i++) {
    wrap_nvmlPciInfo_t pciinfo;
    nvmlh->nvmlDeviceGetPciInfo(nvmlh->devs[i], &pciinfo);
    nvmlh->nvml_pci_domain_id[i] = pciinfo.domain;
    nvmlh->nvml_pci_bus_id[i]    = pciinfo.bus;
    nvmlh->nvml_pci_device_id[i] = pciinfo.device;
  }

  /* build mapping of NVML device IDs to CUDA IDs */
  for (i=0; i<nvmlh->nvml_gpucount; i++) {
    nvmlh->nvml_cuda_device_id[i] = -1;
  } 
  for (i=0; i<nvmlh->cuda_gpucount; i++) {
    hipDeviceProp_t props;
    nvmlh->cuda_nvml_device_id[i] = -1;

    if (hipGetDeviceProperties(&props, i) == hipSuccess) {
      int j;
      for (j=0; j<nvmlh->nvml_gpucount; j++) {
        if ((nvmlh->nvml_pci_domain_id[j] == props.pciDomainID) &&
            (nvmlh->nvml_pci_bus_id[j]    == props.pciBusID) &&
            (nvmlh->nvml_pci_device_id[j] == props.pciDeviceID)) {
#if 0
          printf("CUDA GPU[%d] matches NVML GPU[%d]\n", i, j);
#endif
          nvmlh->nvml_cuda_device_id[j] = i;
          nvmlh->cuda_nvml_device_id[i] = j;
        }
      }
    }
  }

  return nvmlh;
}


int wrap_nvml_destroy(wrap_nvml_handle *nvmlh) {
  nvmlh->nvmlShutdown();

  wrap_dlclose(nvmlh->nvml_dll);
  free(nvmlh);
  return 0;
}


int wrap_nvml_get_gpucount(wrap_nvml_handle *nvmlh, int *gpucount) {
  *gpucount = nvmlh->nvml_gpucount;
  return 0; 
}

int wrap_cuda_get_gpucount(wrap_nvml_handle *nvmlh, int *gpucount) {
  *gpucount = nvmlh->cuda_gpucount;
  return 0; 
}

int wrap_nvml_get_gpu_name(wrap_nvml_handle *nvmlh,
                           int cudaindex, 
                           char *namebuf,
                           int bufsize) {
  int gpuindex = nvmlh->cuda_nvml_device_id[cudaindex];
  if (gpuindex < 0 || gpuindex >= nvmlh->nvml_gpucount)
    return -1;

  if (nvmlh->nvmlDeviceGetName(nvmlh->devs[gpuindex], namebuf, bufsize) != WRAPNVML_SUCCESS)
    return -1; 

  return 0;
}


int wrap_nvml_get_tempC(wrap_nvml_handle *nvmlh,
                        int cudaindex, unsigned int *tempC) {
  wrap_nvmlReturn_t rc;
  int gpuindex = nvmlh->cuda_nvml_device_id[cudaindex];
  if (gpuindex < 0 || gpuindex >= nvmlh->nvml_gpucount)
    return -1;

  rc = nvmlh->nvmlDeviceGetTemperature(nvmlh->devs[gpuindex], 0u /* NVML_TEMPERATURE_GPU */, tempC);
  if (rc != WRAPNVML_SUCCESS) {
    return -1; 
  }

  return 0;
}


int wrap_nvml_get_fanpcnt(wrap_nvml_handle *nvmlh,
                          int cudaindex, unsigned int *fanpcnt) {
  wrap_nvmlReturn_t rc;
  int gpuindex = nvmlh->cuda_nvml_device_id[cudaindex];
  if (gpuindex < 0 || gpuindex >= nvmlh->nvml_gpucount)
    return -1;

  rc = nvmlh->nvmlDeviceGetFanSpeed(nvmlh->devs[gpuindex], fanpcnt);
  if (rc != WRAPNVML_SUCCESS) {
    return -1; 
  }

  return 0;
}


int wrap_nvml_get_power_usage(wrap_nvml_handle *nvmlh,
                              int cudaindex,
                              unsigned int *milliwatts) {
  int gpuindex = nvmlh->cuda_nvml_device_id[cudaindex];
  if (gpuindex < 0 || gpuindex >= nvmlh->nvml_gpucount)
    return -1;

  if (nvmlh->nvmlDeviceGetPowerUsage(nvmlh->devs[gpuindex], milliwatts) != WRAPNVML_SUCCESS)
    return -1; 

  return 0;
}


#if defined(__cplusplus)
}
#endif


